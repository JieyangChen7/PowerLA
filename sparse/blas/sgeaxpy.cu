#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from sparse/blas/zgeaxpy.cu, normal z -> s, Thu Oct  8 23:05:46 2020

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
sgeaxpy_kernel( 
    int num_rows, 
    int num_cols, 
    float alpha, 
    float * dx, 
    float beta, 
    float * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            int idx = row + j*num_rows;
            dy[ idx ] = alpha * dx[ idx ] + beta * dy[ idx ];
        }
    }
}

/**
    Purpose
    -------
    
    This routine computes Y = alpha *  X + beta * Y on the GPU.
    The input format is magma_s_matrix. It can handle both,
    dense matrix (vector block) and CSR matrices. For the latter,
    it interfaces the cuSPARSE library.
    
    Arguments
    ---------

    @param[in]
    alpha       float
                scalar multiplier.
                
    @param[in]
    X           magma_s_matrix
                input/output matrix Y.
                
    @param[in]
    beta        float
                scalar multiplier.
                
    @param[in,out]
    Y           magma_s_matrix*
                input matrix X.
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_sgeaxpy(
    float alpha,
    magma_s_matrix X,
    float beta,
    magma_s_matrix *Y,
    magma_queue_t queue )
{
    int m = X.num_rows;
    int n = X.num_cols;
    magma_s_matrix C={Magma_CSR};
    
    if( X.storage_type == Magma_DENSE && Y->storage_type == Magma_DENSE ){
        
        dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
        magma_int_t threads = BLOCK_SIZE;
        sgeaxpy_kernel<<< grid, threads, 0, queue->hip_stream() >>>
                        ( m, n, alpha, X.dval, beta, Y->dval );
                        
    } else if( X.storage_type == Magma_CSR && Y->storage_type == Magma_CSR ) {
        
        magma_scuspaxpy( &alpha, X, &beta, *Y, &C, queue );
        magma_smfree( Y, queue );
        magma_smtransfer( C, Y, Magma_DEV, Magma_DEV, queue );
        magma_smfree( &C, queue );
    } else {
        printf("%% error: matrix addition only supported for DENSE and CSR format.\n");   
    }
                    
    return MAGMA_SUCCESS;
}
