#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from sparse/blas/magma_zthrsrm.cu, normal z -> s, Thu Oct  8 23:05:48 2020
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#include <hip/hip_runtime.h>

#define SWAP(a, b)  { tmp = a; a = b; b = tmp; }
#define BLOCK_SIZE 128


__global__ void
zcompute_newval_kernel(
    magma_int_t num_rows,
    magma_index_t* Arow,
    magma_index_t* Brow,
    magma_index_t* Acol,
    magma_index_t* Browidx,
    magma_index_t* Bcol,
    float* Aval,
    float* Bval)
{
    int tidx = blockIdx.x*blockDim.x+threadIdx.x;
    magma_index_t offset_new, offset_old, end_old;
    
    if (tidx < num_rows) {
        magma_int_t count = 0;
    
        offset_old = Arow[tidx];
        offset_new = Brow[tidx];
        end_old = Arow[tidx+1];
        
        for (int i = offset_old; i < end_old; i++) {
            if(Acol[i]>-1){
                Bcol[offset_new+count] = Acol[i];
                Bval[offset_new+count] = Aval[i];
                Browidx[offset_new + count] = tidx;
                count++;
            }
        }
    }
}

//kernel
__global__ void
zcompute_nnz_kernel(
    magma_int_t num_rows,
    magma_index_t* Arow,
    magma_index_t* Brow,
    magma_index_t* Acol,
    float* Aval,
    float thrs)
{
    int row= blockIdx.x*blockDim.x+threadIdx.x;
    if (row < num_rows) {
        magma_int_t rm = 0;
        magma_int_t el = 0;
        
        for (int i = Arow[row]; i<Arow[row+1]; i++) {
            if (MAGMA_S_ABS(Aval[i]) <= thrs ) {
                if (Acol[i] != row) {
                    Acol[i] = -1;//cheaperthanval
                    rm++;
                } else {
                    el++;
                }
            } else {
                el++;
            }
        }
        Brow[row] = el;
    }
}



/**
    Purpose
-------
    
    This routine selects a threshold separating the subset_size smallest
    magnitude elements from the rest.
    
    Arguments
    ---------
                
    @param[in]
    order       magma_int_t 
                dummy variable for now.
                
    @param[in,out]
    A           magma_s_matrix*  
                input/output matrix where elements are removed

    @param[out]
    thrs        float*  
                computed threshold

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_saux
    ********************************************************************/
extern "C" magma_int_t
magma_sthrsholdrm_gpu(
    magma_int_t order,
    magma_s_matrix* A,
    float* thrs,
    magma_queue_t queue)
{
    magma_int_t info = 0;
    
    magma_int_t num_blocks = magma_ceildiv(A->num_rows,BLOCK_SIZE);
    magma_s_matrix B={Magma_CSR};
    B.num_rows = A->num_rows;
    B.num_cols = A->num_cols;
    B.storage_type = A->storage_type;
    B.memory_location = Magma_DEV;
    
    magma_index_t *new_rownnz={NULL};

    dim3 block(BLOCK_SIZE, 1, 1);
    dim3 grid(num_blocks, 1, 1 );
    
    magma_index_malloc(&new_rownnz,A->num_rows);
    magma_index_malloc(&B.drow,A->num_rows+1);

    zcompute_nnz_kernel<<<grid, block, 0, queue->hip_stream()>>>
        (A->num_rows, A->drow, new_rownnz, A->dcol, A->dval,*thrs);

    magma_sget_row_ptr(A->num_rows, &B.nnz, new_rownnz, B.drow, queue); 
    magma_smalloc(&B.dval,B.nnz);
    magma_index_malloc(&B.rowidx,B.nnz);
    magma_index_malloc(&B.dcol,B.nnz);
    zcompute_newval_kernel<<<grid, block, 0, queue->hip_stream()>>>
        (A->num_rows, A->drow, B.drow, A->dcol,B.drowidx, B.dcol, A->dval, B.dval); 
   
    //Rewrite the matrix with all the new values
    magma_smatrix_swap(&B, A, queue);
    
    magma_smfree(&B, queue);
    magma_free(new_rownnz);
    return info;
}
