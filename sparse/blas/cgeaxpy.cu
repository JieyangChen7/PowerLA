#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from sparse/blas/zgeaxpy.cu, normal z -> c, Thu Oct  8 23:05:46 2020

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


// axpy kernel for matrices stored in the MAGMA format
__global__ void 
cgeaxpy_kernel( 
    int num_rows, 
    int num_cols, 
    magmaFloatComplex alpha, 
    magmaFloatComplex * dx, 
    magmaFloatComplex beta, 
    magmaFloatComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if( row<num_rows ){
        for( j=0; j<num_cols; j++ ){
            int idx = row + j*num_rows;
            dy[ idx ] = alpha * dx[ idx ] + beta * dy[ idx ];
        }
    }
}

/**
    Purpose
    -------
    
    This routine computes Y = alpha *  X + beta * Y on the GPU.
    The input format is magma_c_matrix. It can handle both,
    dense matrix (vector block) and CSR matrices. For the latter,
    it interfaces the cuSPARSE library.
    
    Arguments
    ---------

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier.
                
    @param[in]
    X           magma_c_matrix
                input/output matrix Y.
                
    @param[in]
    beta        magmaFloatComplex
                scalar multiplier.
                
    @param[in,out]
    Y           magma_c_matrix*
                input matrix X.
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" 
magma_int_t
magma_cgeaxpy(
    magmaFloatComplex alpha,
    magma_c_matrix X,
    magmaFloatComplex beta,
    magma_c_matrix *Y,
    magma_queue_t queue )
{
    int m = X.num_rows;
    int n = X.num_cols;
    magma_c_matrix C={Magma_CSR};
    
    if( X.storage_type == Magma_DENSE && Y->storage_type == Magma_DENSE ){
        
        dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
        magma_int_t threads = BLOCK_SIZE;
        cgeaxpy_kernel<<< grid, threads, 0, queue->hip_stream() >>>
                        ( m, n, alpha, X.dval, beta, Y->dval );
                        
    } else if( X.storage_type == Magma_CSR && Y->storage_type == Magma_CSR ) {
        
        magma_ccuspaxpy( &alpha, X, &beta, *Y, &C, queue );
        magma_cmfree( Y, queue );
        magma_cmtransfer( C, Y, Magma_DEV, Magma_DEV, queue );
        magma_cmfree( &C, queue );
    } else {
        printf("%% error: matrix addition only supported for DENSE and CSR format.\n");   
    }
                    
    return MAGMA_SUCCESS;
}
