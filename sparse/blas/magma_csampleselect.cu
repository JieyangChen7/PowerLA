#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Tobias Ribizel

       @generated from sparse/blas/magma_zsampleselect.cu, normal z -> c, Thu Oct  8 23:05:49 2020
*/

#include "magma_sampleselect.h"
#include <cstdint>

#define PRECISION_c


namespace magma_sampleselect {

__global__ void compute_abs(const magmaFloatComplex* __restrict__ in, float* __restrict__ out, int32_t size) 
{
    auto idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) {
        return;
    }

    auto v = in[idx];
    out[idx] = real(v) * real(v) + imag(v) * imag(v);
}

} // namespace magma_sampleselect

using namespace magma_sampleselect;

/**
    Purpose
    -------

    This routine selects a threshold separating the subset_size smallest
    magnitude elements from the rest.

    Arguments
    ---------

    @param[in]
    total_size  magma_int_t
                size of array val

    @param[in]
    subset_size magma_int_t
                number of smallest elements to separate

    @param[in]
    val         magmaFloatComplex
                array containing the values

    @param[out]
    thrs        float*
                computed threshold

    @param[inout]
    tmp_ptr     magma_ptr*
                pointer to pointer to temporary storage.
                May be reallocated during execution.

    @param[inout]
    tmp_size    magma_int_t*
                pointer to size of temporary storage.
                May be increased during execution.

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_caux
    ********************************************************************/

extern "C" magma_int_t
magma_csampleselect(
    magma_int_t total_size,
    magma_int_t subset_size,
    magmaFloatComplex *val,
    float *thrs,
    magma_ptr *tmp_ptr,
    magma_int_t *tmp_size,
    magma_queue_t queue )
{    
    magma_int_t info = 0;
    magma_int_t arch = magma_getdevice_arch();

    if( arch >= 350 ) {
        magma_int_t num_blocks = magma_ceildiv(total_size, block_size);
        magma_int_t required_size = sizeof(float) * (total_size * 2 + searchtree_size)
                                    + sizeof(int32_t) * sampleselect_alloc_size(total_size);
        auto realloc_result = realloc_if_necessary(tmp_ptr, tmp_size, required_size);

        float* gputmp1 = (float*)*tmp_ptr;
        float* gputmp2 = gputmp1 + total_size;
        float* gputree = gputmp2 + total_size;
        float* gpuresult = gputree + searchtree_size;
        int32_t* gpuints = (int32_t*)(gpuresult + 1);

        CHECK(realloc_result);

        compute_abs<<<num_blocks, block_size, 0, queue->hip_stream()>>>
            (val, gputmp1, total_size);
        sampleselect<<<1, 1, 0, queue->hip_stream()>>>
            (gputmp1, gputmp2, gputree, gpuints, total_size, subset_size, gpuresult);
        magma_sgetvector(1, gpuresult, 1, thrs, 1, queue );
        *thrs = std::sqrt(*thrs);    
    }
    else {
        printf("error: this functionality needs CUDA architecture >= 3.5\n");
        info = MAGMA_ERR_NOT_SUPPORTED;
    }

cleanup:
    return info;
}

/**
    Purpose
    -------

    This routine selects an approximate threshold separating the subset_size
    smallest magnitude elements from the rest.

    Arguments
    ---------

    @param[in]
    total_size  magma_int_t
                size of array val

    @param[in]
    subset_size magma_int_t
                number of smallest elements to separate

    @param[in]
    val         magmaFloatComplex
                array containing the values

    @param[out]
    thrs        float*
                computed threshold

    @param[inout]
    tmp_ptr     magma_ptr*
                pointer to pointer to temporary storage.
                May be reallocated during execution.

    @param[inout]
    tmp_size    magma_int_t*
                pointer to size of temporary storage.
                May be increased during execution.

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_caux
    ********************************************************************/

extern "C" magma_int_t
magma_csampleselect_approx(
    magma_int_t total_size,
    magma_int_t subset_size,
    magmaFloatComplex *val,
    float *thrs,
    magma_ptr *tmp_ptr,
    magma_int_t *tmp_size,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    auto num_blocks = magma_ceildiv(total_size, block_size);
    auto local_work = (total_size + num_threads - 1) / num_threads;
    auto required_size = sizeof(float) * (total_size + searchtree_size)
             + sizeof(int32_t) * (searchtree_width * (num_grouped_blocks + 1) + 1);
    auto realloc_result = realloc_if_necessary(tmp_ptr, tmp_size, required_size);

    float* gputmp = (float*)*tmp_ptr;
    float* gputree = gputmp + total_size;
    uint32_t* gpubucketidx = (uint32_t*)(gputree + searchtree_size);
    int32_t* gpurankout = (int32_t*)(gpubucketidx + 1);
    int32_t* gpucounts = gpurankout + 1;
    int32_t* gpulocalcounts = gpucounts + searchtree_width;
    uint32_t bucketidx{};

    CHECK(realloc_result);

    compute_abs<<<num_blocks, block_size, 0, queue->hip_stream()>>>
        (val, gputmp, total_size);
    build_searchtree<<<1, sample_size, 0, queue->hip_stream()>>>
        (gputmp, gputree, total_size);
    count_buckets<<<num_grouped_blocks, block_size, 0, queue->hip_stream()>>>
        (gputmp, gputree, gpulocalcounts, total_size, local_work);
    reduce_counts<<<searchtree_width, num_grouped_blocks, 0, queue->hip_stream()>>>
        (gpulocalcounts, gpucounts, num_grouped_blocks);
    sampleselect_findbucket<<<1, searchtree_width / 2, 0, queue->hip_stream()>>>
        (gpucounts, subset_size, gpubucketidx, gpurankout);
    magma_getvector(1, sizeof(uint32_t), gpubucketidx, 1, &bucketidx, 1, queue);
    magma_sgetvector(1, gputree + searchtree_width - 1 + bucketidx, 1, thrs, 1, queue);
    *thrs = std::sqrt(*thrs);

cleanup:
    return info;
}
