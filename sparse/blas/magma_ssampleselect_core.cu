#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Tobias Ribizel

       @generated from sparse/blas/magma_dsampleselect_core.cu, normal d -> s, Thu Oct  8 23:05:55 2020
*/

#include "magma_sampleselect.h"

#define PRECISION_s

namespace magma_sampleselect {

constexpr auto max_value = std::numeric_limits<float>::max();

/*
 * Sorting
 */
__device__ void sort2(float* in, int32_t i, int32_t j, bool odd) {
    auto ei = in[i];
    auto ej = in[j];
    if (odd != (ej < ei)) {
        in[i] = ej;
        in[j] = ei;
    }
}

__device__ void bitonic_sort(float* in) {
    int32_t idx = threadIdx.x;
    // idx has the form | high | low | where /low/ has /round/ bits
    for (int32_t round = 0; round < bitonic_cutoff_log2; ++round) {
        // the lowest bit of /high/ decides the sort order
        bool odd = idx & (1 << round);
        for (int32_t bit = 1 << round; bit != 0; bit >>= 1) {
            // idx has the form | upper | lower | where /lower/ initially
            // has /round/ bits and gradually shrink
            int32_t lower = idx & (bit - 1);
            int32_t upper = idx ^ lower;
            // we then sort the elements | upper | 0/1 | lower |
            int32_t sort_idx = lower | (upper << 1);
#if (__CUDACC_VER_MAJOR__ >= 9)
            if (bit >= warp_size) {
                __syncthreads();
            } else {
                __syncwarp();
            }
#else
            __syncthreads();
#endif
            if (idx < 1 << (bitonic_cutoff_log2 - 1)) {
                sort2(in, sort_idx, sort_idx | bit, odd);
            }
        }
    }
}

__global__ void select_bitonic_basecase(float* __restrict__ in, float* __restrict__ out, int32_t size, int32_t rank) {
    __shared__ float data[bitonic_cutoff];
    int32_t idx = threadIdx.x;
    data[threadIdx.x] = idx < size ? in[idx] : max_value;
    __syncthreads();
    bitonic_sort(data);
    __syncthreads();
    if (idx == 0) {
        *out = data[rank];
    }
}

__device__ int32_t searchtree_traversal(const float* searchtree, float el, uint32_t amask, uint32_t& equal_mask) {
    int32_t i = 0;
    equal_mask = amask;
    auto root_splitter = searchtree[0];
    bool next_smaller = el < root_splitter;
    for (int32_t lvl = 0; lvl < searchtree_height; ++lvl) {
        bool smaller = next_smaller;
        i = 2 * i + 2 - smaller;
        next_smaller = el < searchtree[i];
#if (__CUDACC_VER_MAJOR__ >= 9)
        auto local_mask = __ballot_sync(amask, smaller) ^ (smaller - 1);
#else
        auto local_mask = (__ballot(smaller) & amask) ^ (smaller - 1);
#endif
        equal_mask &= local_mask;
    }
    return i - (searchtree_width - 1);
}

__global__ void build_searchtree(const float* __restrict__ in, float* __restrict__ out, int32_t size) {
    __shared__ float sample_buffer[sample_size];
    __shared__ float leaves[searchtree_width];
    auto idx = threadIdx.x;

    sample_buffer[idx] = in[random_pick_idx(idx, sample_size, size)];
    __syncthreads();
    bitonic_sort(sample_buffer);
    __syncthreads();
    if (idx < searchtree_width) {
        leaves[idx] = sample_buffer[uniform_pick_idx(idx, searchtree_width, sample_size)];
        out[idx + searchtree_width - 1] = leaves[idx];
    }
    __syncthreads();
    if (idx < searchtree_width - 1) {
        out[idx] = leaves[searchtree_entry(idx)];
    }
}

template<typename BucketCallback>
__device__ void ssss_impl(const float* __restrict__ in,
                          const float* __restrict__ tree,
                          int32_t size, int32_t workcount,
                          BucketCallback bucket_cb) {
    __shared__ float local_tree[searchtree_size];

    // load searchtree into shared memory
    blockwise_work_local(searchtree_size, [&](int32_t i) {
        local_tree[i] = tree[i];
    });
    __syncthreads();

    blockwise_work(workcount, size, [&](int32_t idx, uint32_t amask) {
        uint32_t equal_mask{};
        auto bucket_idx = searchtree_traversal(local_tree, in[idx], amask, equal_mask);
        // sum of block-wide counts
        bucket_cb(idx, bucket_idx, amask, equal_mask);
    });
}

template<bool write>
__device__ void count_buckets_impl(const float* __restrict__ in,
                                   const float* __restrict__ tree,
                                   int32_t* __restrict__ counts,
                                   uint32_t* __restrict__ oracles,
                                   int32_t size, int32_t workcount) {
    __shared__ int32_t local_counts[searchtree_width];

    blockwise_work_local(searchtree_width, [&](int32_t i) {
        local_counts[i] = 0;
    });
    __syncthreads();
    ssss_impl(in, tree, size, workcount, [&](int32_t idx, int32_t bucket, uint32_t amask, uint32_t mask) {
        if (write) {
            static_assert(searchtree_height <= 8, "can't pack bucket idx into byte");
            store_packed_bytes(oracles, amask, bucket, idx);
        }
        atomicAdd(&local_counts[bucket], 1);
    });
    __syncthreads();
    // store the local counts grouped by block idx
    blockwise_work_local(searchtree_width, [&](int32_t i) {
        counts[i + blockIdx.x * searchtree_width] = local_counts[i];
    });
}

__global__ void count_buckets(const float* __restrict__ in,
                              const float* __restrict__ tree,
                              int32_t* __restrict__ counts,
                              int32_t size, int32_t workcount) {
    count_buckets_impl<false>(in, tree, counts, nullptr, size, workcount);
}

__global__ void count_buckets_write(const float* __restrict__ in,
                                    const float* __restrict__ tree,
                                    int32_t* __restrict__ counts,
                                    uint32_t* __restrict__ oracles,
                                    int32_t size, int32_t workcount) {
    count_buckets_impl<true>(in, tree, counts, oracles, size, workcount);
}

__global__ void collect_bucket_indirect(const float* __restrict__ data,
                                        const uint32_t* __restrict__ oracles_packed,
                                        const int32_t* __restrict__ prefix_sum,
                                        float* __restrict__ out,
                                        int32_t size, uint32_t* bucket_ptr,
                                        int32_t* __restrict__ atomic, int32_t workcount) {
    __shared__ int32_t count;
    auto bucket = *bucket_ptr;
    if (threadIdx.x == 0) {
        count = prefix_sum[bucket + searchtree_width * blockIdx.x];
    }
    __syncthreads();
    blockwise_work(workcount, size, [&](int32_t idx, uint32_t amask) {
            auto packed = load_packed_bytes(oracles_packed, amask, idx);
            int32_t ofs{};
            ofs = warp_aggr_atomic_count_predicate(&count, amask, packed == bucket);
            if (packed == bucket) {
                out[ofs] = data[idx];
            }
        });
}

__device__ void launch_sampleselect(float* __restrict__ in, float* __restrict__ tmp, float* __restrict__ tree,
                                    float* __restrict__ out, int32_t* __restrict__ count_tmp, int32_t size, int32_t rank) {
#if (__CUDA_ARCH >= 350)
    if (threadIdx.x != 0) {
        return;
    }

    if (size <= bitonic_cutoff) {
        select_bitonic_basecase<<<1, bitonic_cutoff>>>(in, out, size, rank);
        return;
    }

    // launch kernels:
    // sample and build searchtree
    build_searchtree<<<1, sample_size>>>(in, tree, size);

    auto local_work = (size + num_threads - 1) / num_threads;
    auto bucket_idx = (uint32_t*)count_tmp;
    auto rank_out = ((int32_t*)bucket_idx) + 1;
    auto atomic = rank_out + 1;
    auto totalcounts = atomic + 1;
    auto localcounts = totalcounts + searchtree_width;
    auto oracles = (uint32_t*)(localcounts + num_grouped_blocks * searchtree_width);

    // count buckets
    count_buckets_write<<<num_grouped_blocks, block_size>>>(in, tree, localcounts, oracles, size, local_work);
    prefix_sum_counts<<<searchtree_width, num_grouped_blocks>>>(localcounts, totalcounts, num_grouped_blocks);
    sampleselect_findbucket<<<1, searchtree_width / 2>>>(totalcounts, rank, bucket_idx, rank_out);
    collect_bucket_indirect<<<num_grouped_blocks, block_size>>>(in, oracles, localcounts, tmp, size, bucket_idx, nullptr, local_work);
    sampleselect_tailcall<<<1, 1>>>(tmp, in, tree, count_tmp, out);
#endif
}

__global__ void sampleselect_tailcall(float* __restrict__ in, float* __restrict__ tmp, float* __restrict__ tree,
                                      int32_t* __restrict__ count_tmp, float* __restrict__ out) {
    if (threadIdx.x != 0) {
        return;
    }
    auto bucket_idx = count_tmp;
    auto rank_out = bucket_idx + 1;
    auto atomic = rank_out + 1;
    auto totalcounts = atomic + 1;

    auto size = totalcounts[*bucket_idx];
    auto rank = *rank_out;
    launch_sampleselect(in, tmp, tree, out, count_tmp, size, rank);
}

__global__ void sampleselect(float* __restrict__ in, float* __restrict__ tmp, float* __restrict__ tree,
                             int32_t* __restrict__ count_tmp, int32_t size, int32_t rank, float* __restrict__ out) {
    launch_sampleselect(in, tmp, tree, out, count_tmp, size, rank);
}

} // namespace magma_sampleselect
