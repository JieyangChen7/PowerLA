#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @precisions mixed zc -> ds

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 256


// CSR-SpMV kernel
__global__ void 
zcgecsrmv_mixed_prec_kernel( 
    int num_rows, 
    int num_cols, 
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * ddiagval,
    magmaFloatComplex * doffdiagval,
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = ddiagval[ row ] * dx[ row ];
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++){
            magmaDoubleComplex val = 
            MAGMA_Z_MAKE( (double) MAGMA_C_REAL(doffdiagval[ j ]),
                          (double) MAGMA_C_IMAG(doffdiagval[ j ])  );
            dot += val * dx[ dcolind[j] ];
        }
        dy[ row ] =  dot *alpha + beta * dy[ row ];
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    A is a matrix in mixed precision, i.e. the diagonal values are stored in
    high precision, the offdiagonal values in low precision.
    The input format is a CSR (val, row, col) in FloatComplex storing all 
    offdiagonal elements and an array containing the diagonal values in 
    DoubleComplex.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       magmaDoubleComplex
                scalar multiplier

    @param[in]
    ddiagval    magmaDoubleComplex_ptr
                array containing diagonal values of A in DoubleComplex
                
    @param[in]
    doffdiagval magmaFloatComplex_ptr
                array containing offdiag values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar multiplier

    @param[out]
    dy          magmaDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zcgecsrmv_mixed_prec(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr ddiagval,
    magmaFloatComplex_ptr doffdiagval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    zcgecsrmv_mixed_prec_kernel<<< grid, threads, 0, queue->hip_stream() >>>
        (m, n, alpha, ddiagval, doffdiagval, drowptr, dcolind, dx, beta, dy);

    return MAGMA_SUCCESS;
}


