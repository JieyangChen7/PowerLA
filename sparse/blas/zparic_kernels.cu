#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define PRECISION_z


__global__ void 
magma_zparic_csr_kernel(    
    magma_int_t n, 
    magma_int_t nnz, 
    magma_index_t *Arowidx, 
    magma_index_t *Acolidx, 
    const magmaDoubleComplex * __restrict__  A_val,
    magma_index_t *rowptr, 
    magma_index_t *colidx, 
    magmaDoubleComplex *val )
{
    int i, j;
    int k = (blockDim.x * blockIdx.x + threadIdx.x); // % nnz;
    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex s, sp;
    int il, iu, jl, ju;
    if ( k < nnz ) {     
        i = Arowidx[k];
        j = Acolidx[k];
#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s = __ldg( A_val+k );
#else
        s = A_val[k];
#endif
        il = rowptr[i];
        iu = rowptr[j];
        while (il < rowptr[i+1] && iu < rowptr[j+1]) {
            sp = zero;
            jl = colidx[il];
            ju = colidx[iu];
            if (jl < ju)
                il++;
            else if (ju < jl)
                iu++;
            else {
                // we are going to modify this u entry
                sp = val[il] * val[iu];
                s -= sp;
                il++;
                iu++;
            }
        }
        s += sp; // undo the last operation (it must be the last)
        // modify entry
        if (i == j) // diagonal
            val[il-1] = MAGMA_Z_MAKE( sqrt( fabs( MAGMA_Z_REAL(s) )), 0.0 );
        else  //sub-diagonal
            val[il-1] =  s / val[iu-1];
    }
}// kernel 


/**
    Purpose
    -------
    
    This routine iteratively computes an incomplete LU factorization.
    For reference, see:
    E. Chow and A. Patel: "Fine-grained Parallel Incomplete LU Factorization", 
    SIAM Journal on Scientific Computing, 37, C169-C193 (2015). 
    This routine was used in the ISC 2015 paper:
    E. Chow et al.: "Asynchronous Iterative Algorithm for Computing Incomplete
                     Factorizations on GPUs", 
                     ISC High Performance 2015, LNCS 9137, pp. 1-16, 2015.
                     
    The input format of the initial guess matrix A is Magma_CSRCOO,
    A_CSR is CSR or CSRCOO format. 

    Arguments
    ---------

    @param[in]
    A           magma_z_matrix
                input matrix A - initial guess (lower triangular)

    @param[in,out]
    A_CSR       magma_z_matrix
                input/output matrix containing the IC approximation
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.
                
    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zparic_csr( 
    magma_z_matrix A,
    magma_z_matrix A_CSR,
    magma_queue_t queue )
{
    int blocksize1 = 128;
    int blocksize2 = 1;

    int dimgrid1 = magma_ceildiv( A.nnz, blocksize1 );
    int dimgrid2 = 1;
    int dimgrid3 = 1;
    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    
    
    magma_zparic_csr_kernel<<< grid, block, 0, queue->hip_stream() >>>
            ( A.num_rows, A.nnz, 
              A.rowidx, A.col, A.val, 
              A_CSR.row, A_CSR.col,  A_CSR.val );

    return MAGMA_SUCCESS;
}
