#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from magmablas/zgerbt_func_batched.cu, normal z -> c, Thu Oct  8 23:05:36 2020

       @author Adrien Remy
       @author Azzam Haidar
*/
#include "magma_internal.h"
#include "cgerbt.h"

#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64

/***************************************************************************//**
    Purpose
    -------
    CPRBT_MVT compute B = UTB to randomize B

    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     COMPLEX array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V

    @param[in,out]
    db     COMPLEX array, dimension (n)
            The n vector db computed by CGESV_NOPIV_GPU
            On exit db = du*db

    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_cprbt_mtv_batched(
    magma_int_t n,
    magmaFloatComplex *du, magmaFloatComplex **db_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t threads = block_length;
    magma_int_t max_batchCount = queue->get_maxBatch();

    for(int i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid( magma_ceildiv( n, 4*block_length ), ibatch);

        magmablas_capply_transpose_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, du, n, db_array+i, 0);
        magmablas_capply_transpose_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, du, n+n/2, db_array+i, n/2);

        threads = block_length;
        grid = magma_ceildiv( n, 2*block_length );
        magmablas_capply_transpose_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n, du, 0, db_array+i, 0);
    }
}


/***************************************************************************//**
    Purpose
    -------
    CPRBT_MV compute B = VB to obtain the non randomized solution

    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in,out]
    db      COMPLEX array, dimension (n)
            The n vector db computed by CGESV_NOPIV_GPU
            On exit db = dv*db

    @param[in]
    dv      COMPLEX array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V

    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_cprbt_mv_batched(
    magma_int_t n,
    magmaFloatComplex *dv, magmaFloatComplex **db_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t threads = block_length;
    magma_int_t max_batchCount = queue->get_maxBatch();

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid ( magma_ceildiv( n, 2*block_length ), ibatch);
        magmablas_capply_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n, dv, 0, db_array+i, 0);

        threads = block_length;
        grid = magma_ceildiv( n, 4*block_length );
        magmablas_capply_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dv, n, db_array+i, 0);
        magmablas_capply_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dv, n+n/2, db_array+i, n/2);
    }
}


/***************************************************************************//**
    Purpose
    -------
    CPRBT randomize a square general matrix using partial randomized transformation

    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.

    @param[in,out]
    dA      COMPLEX array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V

    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).

    @param[in]
    du      COMPLEX array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U

    @param[in]
    dv      COMPLEX array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V

    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_cprbt_batched(
    magma_int_t n,
    magmaFloatComplex **dA_array, magma_int_t ldda,
    magmaFloatComplex *du, magmaFloatComplex *dv,
    magma_int_t batchCount, magma_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 threads2(block_height, block_width);
    magma_int_t max_batchCount = queue->get_maxBatch();

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid( magma_ceildiv( n, 4*block_height ), magma_ceildiv( n, 4*block_width  ), ibatch );

        magmablas_celementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array+i,            0, ldda, du,   0, dv,   0);
        magmablas_celementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array+i,     ldda*n/2, ldda, du,   0, dv, n/2);
        magmablas_celementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array+i,          n/2, ldda, du, n/2, dv,   0);
        magmablas_celementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array+i, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

        dim3 grid2( magma_ceildiv( n, 2*block_height ), magma_ceildiv( n, 2*block_width  ), ibatch );
        magmablas_celementary_multiplication_kernel_batched<<< grid2, threads2, 0, queue->hip_stream() >>>(n, dA_array+i, 0, ldda, du, -ldda, dv, -ldda);
    }
}
