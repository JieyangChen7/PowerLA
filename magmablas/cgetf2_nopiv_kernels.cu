#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Azzam Haidar
       @author Ahmad Abdelfattah

       @generated from magmablas/zgetf2_nopiv_kernels.cu, normal z -> c, Thu Oct  8 23:05:37 2020
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "sync.cuh"
#include "shuffle.cuh"
#include "batched_kernel_param.h"

// This kernel uses registers for matrix storage, shared mem. for communication.
// It also uses lazy swap.
extern __shared__ magmaFloatComplex zdata[];

template<int N>
__device__ void
cgetf2_nopiv_device(int m, magmaFloatComplex* dA, int ldda, magma_int_t *info, const int tx, magmaFloatComplex* sx, int gbstep)
{
    magmaFloatComplex rA[N] = {MAGMA_C_ZERO};
    magmaFloatComplex reg = MAGMA_C_ZERO; 
    
    int linfo = 0;
    float abs;
    // check from previous calls if the panel factorization failed previously
    // this is necessary to report the correct info value 
    if(gbstep > 0 && *info != 0) return;

    // read 
    #pragma unroll
    for(int i = 0; i < N; i++){
        rA[i] = dA[ i * ldda + tx ];
    }
        
    #pragma unroll
    for(int i = 0; i < N; i++){
        if(tx == i){
            #pragma unroll
            for(int j = 0; j < N; j++)
                sx[j] = rA[j];
        }
        __syncthreads();

        abs = fabs(MAGMA_C_REAL( sx[i] )) + fabs(MAGMA_C_IMAG( sx[i] ));
        linfo = ( abs == MAGMA_D_ZERO && linfo == 0) ? (gbstep+i+1) : linfo;
        //linfo = ( abs  == MAGMA_D_ZERO ) ? min(linfo,gbstep+i+1):0;
        reg   = (linfo == 0 ) ? MAGMA_C_DIV(MAGMA_C_ONE, sx[i] ) : MAGMA_C_ONE;

        // scal and ger
        if( tx > i ){
            rA[i] *= reg;
            #pragma unroll
            for(int j = i+1; j < N; j++){
                rA[j] -= rA[i] * sx[j];
            }
        }
        __syncthreads();
    }

    if(tx == 0){
        (*info) = (magma_int_t)( linfo );
    }

    // write
    #pragma unroll
    for(int i = 0; i < N; i++){
        dA[ i * ldda + tx ] = rA[i];
    }
}

/******************************************************************************/
extern __shared__ magmaFloatComplex zdata[];
template<int N, int NPOW2>
__global__ void
cgetf2_nopiv_batched_kernel( int m, magmaFloatComplex** dA_array, int ai, int aj, int ldda, 
                             magma_int_t* info_array, int gbstep, int batchCount)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int batchid = blockIdx.x * blockDim.y + ty;
    if(batchid >= batchCount)return;

    magmaFloatComplex* dA = dA_array[batchid] + aj * ldda + ai;
    magma_int_t* info = &info_array[batchid];
    magmaFloatComplex* sx = (magmaFloatComplex*)zdata;
    sx += ty * NPOW2;

    cgetf2_nopiv_device<N>(m, dA, ldda, info, tx, sx, gbstep);
}
/***************************************************************************//**
    Purpose
    -------
    cgetf2_nopiv computes the non-pivoting LU factorization of an M-by-N matrix A.
    This routine can deal with matrices of limited widths, so it is for internal use.

    The factorization has the form
       A = L * U
    where L is lower triangular with unit diagonal elements (lower
    trapezoidal if m > n), and U is upper triangular (upper
    trapezoidal if m < n).

    This is a batched version that factors batchCount M-by-N matrices in parallel.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows the matrix A.  N >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in,out]
    dA_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX array on the GPU, dimension (LDDA,N).
            On entry, each pointer is an M-by-N matrix to be factored.
            On exit, the factors L and U from the factorization
            A = L*U; the unit diagonal elements of L are not stored.

    @param[in]
    ai      INTEGER
            Row offset for dA_array.

    @param[in]
    aj      INTEGER
            Column offset for dA_array.

    @param[in]
    ldda    INTEGER
            The leading dimension of each array A.  LDDA >= max(1,M).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.
      -     > 0:  if INFO = i, U(i,i) is exactly zero. The factorization
                  has been completed, but the factor U is exactly
                  singular, and division by zero will occur if it is used
                  to solve a system of equations.

    @param[in]
    gbstep      INTEGER
                Internal use.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_getrf_batched
*******************************************************************************/
extern "C" magma_int_t 
magma_cgetf2_nopiv_internal_batched( 
    magma_int_t m, magma_int_t n, 
    magmaFloatComplex** dA_array, magma_int_t ai, magma_int_t aj, magma_int_t ldda, 
    magma_int_t* info_array, magma_int_t gbstep, 
    magma_int_t batchCount, magma_queue_t queue )
{
    #define dAarray(i,j) dA_array, i, j

    magma_int_t arginfo = 0;
    if (m < 0) {
        arginfo = -1;
    } else if (n < 0 || n > 32 || (m > 512 && n > 16) ) {
        arginfo = -2;
    } else if (ai < 0) {
        arginfo = -4;
    } else if (aj < 0) {
        arginfo = -5;
    } else if (ldda < max(1,m)) {
        arginfo = -6;
    }

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }

    // Quick return if possible
    if (m == 0 || n == 0) {
        return arginfo;
    }

    magma_int_t m1 = (m > MAX_NTHREADS) ? MAX_NTHREADS : m;
    magma_int_t m2 = m - m1;

    const magma_int_t ntcol = (m1 > 32) ? 1 : (2 * (32/m1));
    magma_int_t shmem = ntcol * magma_ceilpow2(n) * sizeof(magmaFloatComplex);
    magma_int_t gridx = magma_ceildiv(batchCount, ntcol);
    dim3 threads(m1, ntcol, 1);
    dim3 grid(gridx, 1, 1);
    switch(n){
        case  1: cgetf2_nopiv_batched_kernel< 1, magma_ceilpow2( 1)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  2: cgetf2_nopiv_batched_kernel< 2, magma_ceilpow2( 2)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  3: cgetf2_nopiv_batched_kernel< 3, magma_ceilpow2( 3)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  4: cgetf2_nopiv_batched_kernel< 4, magma_ceilpow2( 4)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  5: cgetf2_nopiv_batched_kernel< 5, magma_ceilpow2( 5)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  6: cgetf2_nopiv_batched_kernel< 6, magma_ceilpow2( 6)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  7: cgetf2_nopiv_batched_kernel< 7, magma_ceilpow2( 7)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  8: cgetf2_nopiv_batched_kernel< 8, magma_ceilpow2( 8)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case  9: cgetf2_nopiv_batched_kernel< 9, magma_ceilpow2( 9)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 10: cgetf2_nopiv_batched_kernel<10, magma_ceilpow2(10)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 11: cgetf2_nopiv_batched_kernel<11, magma_ceilpow2(11)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 12: cgetf2_nopiv_batched_kernel<12, magma_ceilpow2(12)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 13: cgetf2_nopiv_batched_kernel<13, magma_ceilpow2(13)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 14: cgetf2_nopiv_batched_kernel<14, magma_ceilpow2(14)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 15: cgetf2_nopiv_batched_kernel<15, magma_ceilpow2(15)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 16: cgetf2_nopiv_batched_kernel<16, magma_ceilpow2(16)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 17: cgetf2_nopiv_batched_kernel<17, magma_ceilpow2(17)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 18: cgetf2_nopiv_batched_kernel<18, magma_ceilpow2(18)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 19: cgetf2_nopiv_batched_kernel<19, magma_ceilpow2(19)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 20: cgetf2_nopiv_batched_kernel<20, magma_ceilpow2(20)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 21: cgetf2_nopiv_batched_kernel<21, magma_ceilpow2(21)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 22: cgetf2_nopiv_batched_kernel<22, magma_ceilpow2(22)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 23: cgetf2_nopiv_batched_kernel<23, magma_ceilpow2(23)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 24: cgetf2_nopiv_batched_kernel<24, magma_ceilpow2(24)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 25: cgetf2_nopiv_batched_kernel<25, magma_ceilpow2(25)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 26: cgetf2_nopiv_batched_kernel<26, magma_ceilpow2(26)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 27: cgetf2_nopiv_batched_kernel<27, magma_ceilpow2(27)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 28: cgetf2_nopiv_batched_kernel<28, magma_ceilpow2(28)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 29: cgetf2_nopiv_batched_kernel<29, magma_ceilpow2(29)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 30: cgetf2_nopiv_batched_kernel<30, magma_ceilpow2(30)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 31: cgetf2_nopiv_batched_kernel<31, magma_ceilpow2(31)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        case 32: cgetf2_nopiv_batched_kernel<32, magma_ceilpow2(32)><<<grid, threads, shmem, queue->hip_stream()>>>(m1, dA_array, ai, aj, ldda, info_array, gbstep, batchCount); break;
        default: printf("error: panel width %lld is not supported\n", (long long) n);
    }

    if(m2 > 0){
        magmablas_ctrsm_recursive_batched( 
            MagmaRight, MagmaUpper, MagmaNoTrans, MagmaNonUnit, 
            m2, n, MAGMA_C_ONE, 
            dAarray(ai   ,aj), ldda, 
            dAarray(ai+m1,aj), ldda, batchCount, queue );
    }

    #undef dAarray
    return arginfo;
}
