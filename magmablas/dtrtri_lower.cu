#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from magmablas/ztrtri_lower.cu, normal z -> d, Thu Oct  8 23:05:35 2020

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       This file implements lower case, and is called by dtrtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "dtrtri.cuh"
#include "dtrtri_lower_device.cuh"


/******************************************************************************/
__global__ void
dtrtri_diag_lower_kernel(
    magma_diag_t diag, int n, const double *A, int lda, double *d_dinvA)
{
    dtrtri_diag_lower_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_dgemm16_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm16_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm16_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm16_part2_lower_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm32_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm32_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm32_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm32_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm64_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm64_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm_above64_part1_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm_above64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm_above64_part2_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm_above64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_dgemm_above64_part3_lower_kernel(
    int n, const double *Ain, int lda, double *d_dinvA, int jb, int npages)
{
    triple_dgemm_above64_part3_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}
