#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Ahmad Abdelfattah
       @author Azzam Haidar

       @generated from magmablas/zgemm_batched_smallsq.cu, normal z -> c, Thu Oct  8 23:05:36 2020
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

#define SLDA(N)    ( (N==15||N==23||N==31)? N : (N+1) )

extern __shared__ magmaFloatComplex zdata[];
template<int N>
__global__ void
cgemm_batched_smallsq_kernel(
        const magma_trans_t transA, magma_trans_t transB, 
        const magmaFloatComplex alpha, magmaFloatComplex const * const * dA_array, int ai, int aj, int ldda, 
                                        magmaFloatComplex const * const * dB_array, int bi, int bj, int lddb, 
        const magmaFloatComplex beta,  magmaFloatComplex**               dC_array, int ci, int cj, int lddc, 
        const int batchCount)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    const int bx = blockIdx.x;
    
    const int batchid = bx * blockDim.z + tz;
    if(batchid >= batchCount) return;
    
    const magmaFloatComplex* __restrict__ dA = dA_array[batchid] + aj * ldda + ai;
    const magmaFloatComplex* __restrict__ dB = dB_array[batchid] + bj * lddb + bi;
          magmaFloatComplex* __restrict__ dC = dC_array[batchid] + cj * lddc + ci;
    
    magmaFloatComplex rC = MAGMA_C_ZERO; 
    magmaFloatComplex rTmp = MAGMA_C_ZERO; 
    
    const int slda = SLDA(N);
    const int sldb = SLDA(N);
    magmaFloatComplex* sA = (magmaFloatComplex*)(zdata);
    magmaFloatComplex* sB = (magmaFloatComplex*)(zdata + blockDim.z * slda * N);
    
    sA += tz * slda * N;
    sB += tz * sldb * N;
    
    // read A & B 
    if(transA == MagmaNoTrans){
        sA[ty * slda + tx] = dA[ty * ldda + tx];
    }
    else{
        sA[tx * slda + ty] = (transA == MagmaTrans) ? dA[ty * ldda + tx] : MAGMA_C_CONJ( dA[ty * ldda + tx] );
    }

    if(transB == MagmaNoTrans){
        sB[ty * sldb + tx] = dB[ty * lddb + tx];
    }
    else{
        sB[tx * sldb + ty] = (transB == MagmaTrans) ? dB[ty * lddb + tx] : MAGMA_C_CONJ( dB[ty * lddb + tx] );
    }
    __syncthreads(); 

    if(beta != MAGMA_C_ZERO){
        rC = beta * dC[ty * lddc + tx];
    }

    // multiply
    rTmp = MAGMA_C_ZERO;
    #pragma unroll
    for(int j = 0; j < N; j++){
        rTmp += sA[j * slda + tx] * sB[ty * sldb + j]; 
    }
    rC += alpha * rTmp;

    // write from rC
    dC[ty * lddc + tx] = rC;
}


extern "C" void 
magmablas_cgemm_batched_smallsq(
    magma_trans_t transA, magma_trans_t transB, 
    magma_int_t m, magma_int_t n, magma_int_t k, 
    magmaFloatComplex alpha,
    magmaFloatComplex const * const * dA_array, magma_int_t ai, magma_int_t aj, magma_int_t ldda, 
    magmaFloatComplex const * const * dB_array, magma_int_t bi, magma_int_t bj, magma_int_t lddb,
    magmaFloatComplex beta,
    magmaFloatComplex **dC_array, magma_int_t ci, magma_int_t cj, magma_int_t lddc, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    if      ( transA != MagmaNoTrans && transA != MagmaTrans && transA != MagmaConjTrans )
        info = -1;
    else if ( transB != MagmaNoTrans && transB != MagmaTrans && transB != MagmaConjTrans )
        info = -2;
    else if ( m < 0 )
        info = -3;
    else if ( n < 0 )
        info = -4;
    else if ( k < 0 )
        info = -5;
    else if ( transA == MagmaNoTrans ? ldda < m : ldda < k )
        info = -8;
    else if ( transB == MagmaNoTrans ? lddb < k : lddb < n )
        info = -10;
    else if ( lddc < m )
        info = -13;
    
    if( !(m == n  && n == k) ){
        printf("Only square sizes are supported\n");
        info = -1;
    }

    if( m > 32){
        printf("Only square sizes of up to 32 are supported\n");
        info = -1;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
    }

    if ( m <= 0 || n <= 0 || k <= 0 ) return;
    
    magma_int_t ntcol  = magma_get_cgemm_batched_ntcol( m );
    magma_int_t shmem  = ( SLDA(m)*m + SLDA(n)*n ) * sizeof(magmaFloatComplex);
                shmem *= ntcol;

    const int nblocks = magma_ceildiv(batchCount, ntcol);
    dim3 grid(nblocks, 1, 1);
    dim3 threads(m, m, ntcol);

    switch(m){
        case  1: cgemm_batched_smallsq_kernel< 1><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  2: cgemm_batched_smallsq_kernel< 2><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  3: cgemm_batched_smallsq_kernel< 3><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  4: cgemm_batched_smallsq_kernel< 4><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  5: cgemm_batched_smallsq_kernel< 5><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  6: cgemm_batched_smallsq_kernel< 6><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  7: cgemm_batched_smallsq_kernel< 7><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  8: cgemm_batched_smallsq_kernel< 8><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  9: cgemm_batched_smallsq_kernel< 9><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 10: cgemm_batched_smallsq_kernel<10><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 11: cgemm_batched_smallsq_kernel<11><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 12: cgemm_batched_smallsq_kernel<12><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 13: cgemm_batched_smallsq_kernel<13><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 14: cgemm_batched_smallsq_kernel<14><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 15: cgemm_batched_smallsq_kernel<15><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 16: cgemm_batched_smallsq_kernel<16><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 17: cgemm_batched_smallsq_kernel<17><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 18: cgemm_batched_smallsq_kernel<18><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 19: cgemm_batched_smallsq_kernel<19><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 20: cgemm_batched_smallsq_kernel<20><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 21: cgemm_batched_smallsq_kernel<21><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 22: cgemm_batched_smallsq_kernel<22><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 23: cgemm_batched_smallsq_kernel<23><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 24: cgemm_batched_smallsq_kernel<24><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 25: cgemm_batched_smallsq_kernel<25><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 26: cgemm_batched_smallsq_kernel<26><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 27: cgemm_batched_smallsq_kernel<27><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 28: cgemm_batched_smallsq_kernel<28><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 29: cgemm_batched_smallsq_kernel<29><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 30: cgemm_batched_smallsq_kernel<30><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 31: cgemm_batched_smallsq_kernel<31><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 32: cgemm_batched_smallsq_kernel<32><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        default:;
    }
}
