#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Azzam Haidar
       @author Ahmad Abdelfattah

       @generated from magmablas/zgetf2_native_kernel.cu, normal z -> s, Thu Oct  8 23:05:37 2020
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "shuffle.cuh"
#include "sync.cuh"
#include "atomics.cuh"
#include "batched_kernel_param.h"

#define PRECISION_s

/**
    Purpose
    -------
    LU factorization of m-by-n matrix ( m >= n ).
    Each thread block caches an entire column in register.
    Thread blocks communicate and synchronize through global memory.
    Assumptions:
        1. dA is of size MxN such that N <= M.
        2. Thread block must be 1D, with TX multiple of 32 (warp size)
        3. TX must be >= n
        4. n must be less than the number of SMs on the GPU
**/

// =============================================================================
// init kernel
__global__ void
sgetf2_native_init_kernel( int n, int npages, magma_int_t *ipiv, int* update_flags)
{
    const int tx = threadIdx.x;
    if( tx < n){
        ipiv[ tx ] = 0;
    }
    if( tx < max(n,npages) ){
        update_flags[ tx ] = 0;
    }
}

// =============================================================================
// the main kernel
template<int TX, int NPAGES>
__global__ void
sgetf2_native_kernel( int m, int n,
                      magmaFloat_ptr dA, int ldda,
                      volatile magma_int_t *ipiv, int gbstep,
                      volatile int* update_flag,
                      volatile magma_int_t *info)
{
#ifdef HAVE_CUBLAS
    const int tx  = threadIdx.x;
    const int bx = blockIdx.x;
    float rA[NPAGES] = {MAGMA_S_ZERO};
    float rx, rx_max;
    magmaFloat_ptr da = dA;
    int rx_id, max_id, flag = 0;
    float  rx_abs = 0.0, rx_abs_max = 0.0;
    const int m_ = m-(NPAGES-1)*TX;
    if( bx >= n ) return;

    __shared__ float sx[ TX ];
    __shared__ float sabs[ TX ];
    __shared__ int smax_id[ TX ];
    __shared__ float sreg;

    // read
    dA += bx * ldda + tx;
    #pragma unroll
    for(int i = 0; i < NPAGES-1; i++){
        rA[i] = dA[ i * TX ];
    }
    if( tx <  m_){
        rA[NPAGES-1] = dA[ (NPAGES-1) * TX ];
    }

    // main loop
    #pragma unroll
    for(int i = 0; i < n; i++){
        // isamax and write pivot for the ith thread block
        if(bx == i){
            rx_max     = rx     = (tx < i) ? MAGMA_S_ZERO : rA[0];
            rx_abs_max = rx_abs = fabs(MAGMA_S_REAL(rx)) + fabs(MAGMA_S_IMAG(rx));
            max_id = rx_id = tx;
            #pragma unroll
            for(int j = 1; j < NPAGES; j++){
                rx = rA[j];
                rx_abs = fabs(MAGMA_S_REAL(rx)) + fabs(MAGMA_S_IMAG(rx));
                if ( rx_abs  > rx_abs_max ){
                    rx_max = rx;
                    rx_abs_max = rx_abs;
                    max_id = j * TX + tx;
                }
            }
            sx[ tx ] = rx_max;
            sabs[ tx ] = rx_abs_max;
            smax_id[ tx ] = max_id;
            __syncthreads();

            // let the first warp do the final reduction step
            if(tx < 32){
                #pragma unroll
                for(int j = 0; j < TX; j+= 32){
                    rx     = sx[ j + tx ];
                    rx_abs = sabs[ j + tx ];
                    rx_id  = smax_id[ j + tx ];
                    if ( rx_abs  > rx_abs_max ){
                        rx_max = rx;
                        rx_abs_max = rx_abs;
                        max_id = rx_id;
                    }
                }
                magmablas_syncwarp();
                sx[ tx ] = rx_max;
                sabs[ tx ] = rx_abs_max;
                smax_id[ tx ] = max_id;
                magmablas_syncwarp();
                #pragma unroll
                for(int j = 0; j < 32; j++){
                    rx     = sx[j];
                    rx_abs = sabs[j];
                    rx_id  = smax_id[j];
                    if ( rx_abs  > rx_abs_max ){
                        rx_abs_max = rx_abs;
                        rx_max = rx;
                        max_id = rx_id;
                    }
                }
            }

            if(tx == 0){
                sx[ 0 ] = rx_max;
                sabs[ 0 ] = rx_abs_max;
                smax_id[ 0 ] = max_id;
            }
            __syncthreads();
            rx_max = sx[ 0 ];
            rx_abs_max = sabs[ 0 ];
            max_id = smax_id[ 0 ];
            __syncthreads();

            // now every thread in the i^th block has the maximum
            if( tx == 0){
                if( rx_abs_max == MAGMA_D_ZERO){
                    magmablas_iatomic_exchange( (magma_int_t*)info, (magma_int_t)(max_id + gbstep + 1) );
                }
                magmablas_iatomic_exchange((magma_int_t*)&ipiv[i], (magma_int_t)(max_id+1) ); // fortran indexing
            }
            __syncthreads();
            if( rx_abs_max == MAGMA_D_ZERO )return;
        }
        else{ // other thread blocks are waiting
            if(tx == 0){
                max_id = 0;
                while( max_id == 0 ){
                    max_id = ipiv[i];
                };
                smax_id[ 0 ] = max_id;
            }
            __syncthreads();
            max_id = smax_id[ 0 ];
            max_id -= 1; // revert fortran indexing
            __syncthreads();
            if( (*info) != 0 ) return;
        }

        // swap
        // swap always happens between page 0 and page x
        // to avoid spilling rA to local memory, we use shared memory
        if( max_id != i){
            // all blocks swap in registers
            // for bx < i, the column is already written in memory,
            // but we have a copy in reg., so continue to swap in reg.,
            // and do one final write to memory
            #pragma unroll
            for(int j = 0; j < NPAGES; j++){
                if( j == (max_id/TX) ){
                    sx[ tx ] = rA[j];
                    __syncthreads();
                    if( tx == i ){
                        float tmp    = sx[ max_id%TX ];
                        sx[ max_id%TX ] = rA[0];
                        rA[0] = tmp;
                    }
                    __syncthreads();
                    if( tx == max_id%TX ){
                        rA[j] = sx[ tx ];
                    }
                    __syncthreads();
                }
            }
            //__syncthreads();
        }

        // the ith block does scal
        if(bx == i){
            float reg = MAGMA_S_DIV(MAGMA_S_ONE, rx_max );
            // scal
            if( tx > i ){
                rA[0] *= reg;
            }
            #pragma unroll
            for(int j = 1; j < NPAGES; j++){
                rA[j] *= reg;
            }
            // write column i to global memory
            #pragma unroll
            for(int j = 0; j < NPAGES-1; j++){
                dA[ j * TX ] = rA[j];
            }
            if( tx <  m_){
                dA[ (NPAGES-1) * TX ] = rA[NPAGES-1];
            }
            __threadfence(); __syncthreads(); // after cuda 9.0, both are needed, not sure why
            if(tx == 0) magmablas_iatomic_exchange( (int *)&update_flag[ i ], 1);
        }

        // thread blocks with ID larger than i perform ger
        if(bx > i){
            if( tx == i ){
                sreg = rA[0];
            }
            // wait for scal
            if( tx == 0){
                flag = 0;
                while( flag == 0 ){
                    flag = update_flag[ i ];
                };
            }
            __syncthreads();

            float reg = sreg;
            if( NPAGES == 1){
                if(tx > i && tx < m_){
                    rA[0] -= da[ i * ldda + tx ] * reg;
                }
            }else{
                if(tx > i){
                    rA[0] -= da[ i * ldda + tx ] * reg;
                }
            }
            #pragma unroll
            for(int j = 1; j < NPAGES-1; j++){
                rA[j] -= da[ i * ldda + j * TX + tx ] * reg;
            }
            if( NPAGES > 1){
                if( tx < m_ ){
                    rA[ NPAGES-1 ] -= da[ i * ldda + (NPAGES-1)*TX + tx ] * reg;
                }
            }
        }
    }

    // all blocks write their columns again except the last one
    if( bx < n-1 ){
        #pragma unroll
        for(int i = 0; i < NPAGES-1; i++){
            dA[ i * TX ] = rA[i];
        }
        if( tx <  m_){
            dA[ (NPAGES-1) * TX ] = rA[NPAGES-1];
        }
    }

#endif    // HAVE_CUBLAS
}

// =============================================================================
extern "C" magma_int_t
magma_sgetf2_native_fused(
    magma_int_t m, magma_int_t n,
    magmaFloat_ptr dA, magma_int_t ldda,
    magma_int_t *ipiv, magma_int_t gbstep,
    magma_int_t *flags,
    magma_int_t *info, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    const magma_int_t ntx   = SGETF2_FUSED_NTH;

    if( m < n || m > SGETF2_FUSED_MAX_M ){
        arginfo = -1;
    }
    else if( n > magma_getdevice_multiprocessor_count() ){
        arginfo = -2;
    }
    else if( ldda < max(1, m) ){
        arginfo = -4;
    }

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }

    magma_int_t arch = magma_getdevice_arch();

    dim3 grid(n, 1, 1);
    dim3 threads(ntx, 1, 1);
    const magma_int_t npages = magma_ceildiv(m, ntx);
    // the kernel uses communication among thread blocks
    // as a safeguard, force one thread block per multiprocessor
    // by allocating more than half the shared memory
    magma_int_t shmem = magma_getdevice_shmem_block();
    shmem = (shmem / 2);
    int *update_flag = (int*) flags;    // update_flag is an int, not magma_int_t
    sgetf2_native_init_kernel<<< 1, max(n,npages), 0, queue->hip_stream() >>>( n, npages, ipiv, update_flag);
    // The case statement should cover up to ( xGETF2_CHAIN_MAX_M / ntx )
    switch(npages){
        case  1: sgetf2_native_kernel< ntx,  1><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  2: sgetf2_native_kernel< ntx,  2><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  3: sgetf2_native_kernel< ntx,  3><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  4: sgetf2_native_kernel< ntx,  4><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  5: sgetf2_native_kernel< ntx,  5><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  6: sgetf2_native_kernel< ntx,  6><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  7: sgetf2_native_kernel< ntx,  7><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  8: sgetf2_native_kernel< ntx,  8><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  9: sgetf2_native_kernel< ntx,  9><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 10: sgetf2_native_kernel< ntx, 10><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 11: sgetf2_native_kernel< ntx, 11><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 12: sgetf2_native_kernel< ntx, 12><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 13: sgetf2_native_kernel< ntx, 13><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 14: sgetf2_native_kernel< ntx, 14><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 15: sgetf2_native_kernel< ntx, 15><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 16: sgetf2_native_kernel< ntx, 16><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 17: sgetf2_native_kernel< ntx, 17><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 18: sgetf2_native_kernel< ntx, 18><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 19: sgetf2_native_kernel< ntx, 19><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 20: sgetf2_native_kernel< ntx, 20><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        #if defined(PRECISION_s) || defined(PRECISION_d)
        case 21: sgetf2_native_kernel< ntx, 21><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 22: sgetf2_native_kernel< ntx, 22><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 23: sgetf2_native_kernel< ntx, 23><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 24: sgetf2_native_kernel< ntx, 24><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 25: sgetf2_native_kernel< ntx, 25><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 26: sgetf2_native_kernel< ntx, 26><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 27: sgetf2_native_kernel< ntx, 27><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 28: sgetf2_native_kernel< ntx, 28><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 29: sgetf2_native_kernel< ntx, 29><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 30: sgetf2_native_kernel< ntx, 30><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 31: sgetf2_native_kernel< ntx, 31><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 32: sgetf2_native_kernel< ntx, 32><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 33: sgetf2_native_kernel< ntx, 33><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 34: sgetf2_native_kernel< ntx, 34><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 35: sgetf2_native_kernel< ntx, 35><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 36: sgetf2_native_kernel< ntx, 36><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 37: sgetf2_native_kernel< ntx, 37><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 38: sgetf2_native_kernel< ntx, 38><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 39: sgetf2_native_kernel< ntx, 39><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 40: sgetf2_native_kernel< ntx, 40><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 41: sgetf2_native_kernel< ntx, 41><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 42: sgetf2_native_kernel< ntx, 42><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 43: sgetf2_native_kernel< ntx, 43><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 44: sgetf2_native_kernel< ntx, 44><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 45: sgetf2_native_kernel< ntx, 45><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 46: sgetf2_native_kernel< ntx, 46><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        #endif // defined(PRECISION_s) || defined(PRECISION_d)
        #if defined(PRECISION_s)
        case 47: sgetf2_native_kernel< ntx, 47><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 48: sgetf2_native_kernel< ntx, 48><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 49: sgetf2_native_kernel< ntx, 49><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 50: sgetf2_native_kernel< ntx, 50><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 51: sgetf2_native_kernel< ntx, 51><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 52: sgetf2_native_kernel< ntx, 52><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 53: sgetf2_native_kernel< ntx, 53><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 54: sgetf2_native_kernel< ntx, 54><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 55: sgetf2_native_kernel< ntx, 55><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 56: sgetf2_native_kernel< ntx, 56><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 57: sgetf2_native_kernel< ntx, 57><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 58: sgetf2_native_kernel< ntx, 58><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 59: sgetf2_native_kernel< ntx, 59><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 60: sgetf2_native_kernel< ntx, 60><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 61: sgetf2_native_kernel< ntx, 61><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 62: sgetf2_native_kernel< ntx, 62><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 63: sgetf2_native_kernel< ntx, 63><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 64: sgetf2_native_kernel< ntx, 64><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 65: sgetf2_native_kernel< ntx, 65><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 66: sgetf2_native_kernel< ntx, 66><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 67: sgetf2_native_kernel< ntx, 67><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 68: sgetf2_native_kernel< ntx, 68><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 69: sgetf2_native_kernel< ntx, 69><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 70: sgetf2_native_kernel< ntx, 70><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 71: sgetf2_native_kernel< ntx, 71><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 72: sgetf2_native_kernel< ntx, 72><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 73: sgetf2_native_kernel< ntx, 73><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 74: sgetf2_native_kernel< ntx, 74><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 75: sgetf2_native_kernel< ntx, 75><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 76: sgetf2_native_kernel< ntx, 76><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 77: sgetf2_native_kernel< ntx, 77><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 78: sgetf2_native_kernel< ntx, 78><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 79: sgetf2_native_kernel< ntx, 79><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 80: sgetf2_native_kernel< ntx, 80><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        #endif // defined(PRECISION_s)
        default: printf("size not supported \n");
    }
    return 0;
}
