#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Azzam Haidar
       @author Tingxing Dong
       @author Ahmad Abdelfattah

       @precisions normal z -> s d c
*/

#include "magma_internal.h"
#include "batched_kernel_param.h"
#include "magma_templates.h"
#include "shuffle.cuh"

/*
    Purpose
    -------
    These are internal routines that might have many assumption.
    They are used in zgetf2_batched.cpp
*/


#define PRECISION_z

#define A(i, j)  (A + (i) + (j)*lda)   // A(i, j) means at i row, j column

/******************************************************************************/
extern __shared__ magmaDoubleComplex shared_data[];
extern __shared__ double sdata[];
extern __shared__ int int_sdata[];


/******************************************************************************/
__device__ int
izamax_devfunc(int length, const magmaDoubleComplex *x, int incx, double *shared_x, int *shared_idx)
{
    int tx = threadIdx.x;
    magmaDoubleComplex res;
    double  res1;
    int nchunk = magma_ceildiv( length, zamax );

    if ( tx < zamax ) {
        shared_x[tx]   = 0.0;
        shared_idx[tx] = tx; //-1; // -1 will crash the code in case matrix is singular, better is to put =tx and make check info at output
    }
    __syncthreads();

    for (int s =0; s < nchunk; s++)
    {
        if ( (tx + s * zamax < length) && (tx < zamax) )
        {
            res = x[(tx + s * zamax) * incx];
            res1 = fabs(MAGMA_Z_REAL(res)) + fabs(MAGMA_Z_IMAG(res));

            if ( res1  > shared_x[tx] )
            {
                shared_x[tx] = res1;
                shared_idx[tx] = tx + s * zamax;
            }
        }
    }
    __syncthreads();

    if (length >= zamax) // there are more than 128 threads working ==> all shared_x shared_idx are initialized here so I can call the fixed getidmax
        magma_getidmax<zamax>(tx, shared_x, shared_idx);
    else
        magma_getidmax_n(min(zamax,length), tx, shared_x, shared_idx);
    return shared_idx[0];
}

/******************************************************************************/
__global__ void
izamax_kernel_batched(int length, int chunk, magmaDoubleComplex **x_array, int xi, int xj, int incx,
                   int step, int lda, magma_int_t** ipiv_array, magma_int_t *info_array, int gbstep)
{
    const int batchid = blockIdx.x;
    magmaDoubleComplex *x_start = x_array[batchid] + xj * lda + xi;
    const magmaDoubleComplex *x = &(x_start[step + step * lda]);

    magma_int_t *ipiv = ipiv_array[batchid] + xi;
    int tx = threadIdx.x;

    double *shared_x = sdata;
    int *shared_idx = (int*)(shared_x + zamax);

    izamax_devfunc(length, x, incx, shared_x, shared_idx);

    if (tx == 0) {
        ipiv[step]  = shared_idx[0] + step + 1; // Fortran Indexing
        if (shared_x[0] == MAGMA_D_ZERO) {
            info_array[batchid] = shared_idx[0] + step + gbstep + 1;
        }
    }
}


/******************************************************************************/
__global__ void
izamax_kernel_native(int length, int chunk, magmaDoubleComplex_ptr x, int incx,
                     int step, int lda, magma_int_t* ipiv, magma_int_t *info, int gbstep)
{
    const int tx = threadIdx.x;
    x += step * lda + step;

    double *shared_x = sdata;
    int *shared_idx = (int*)(shared_x + zamax);

    izamax_devfunc(length, x, incx, shared_x, shared_idx);
    if (tx == 0) {
        ipiv[step]  = shared_idx[0] + step + 1; // Fortran Indexing
        if (shared_x[0] == MAGMA_D_ZERO) {
            (*info) = shared_idx[0] + step + gbstep + 1;
        }
    }
}


/***************************************************************************//**
    Purpose
    -------

    IZAMAX find the index of max absolute value of elements in x and store the index in ipiv

    This is an internal routine that might have many assumption.

    Arguments
    ---------

    @param[in]
    length       INTEGER
            On entry, length specifies the size of vector x. length >= 0.


    @param[in]
    x_array     Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array of dimension


    @param[in]
    xi      INTEGER
            Row offset, internal use

    @param[in]
    xj      INTEGER
            Column offset, internal use

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    step    INTEGER
            the offset of ipiv

    @param[in]
    lda    INTEGER
            The leading dimension of each array A, internal use to find the starting position of x.

    @param[out]
    ipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.
      -     > 0:  if INFO = i, U(i,i) is exactly zero. The factorization
                  has been completed, but the factor U is exactly
                  singular, and division by zero will occur if it is used
                  to solve a system of equations.

    @param[in]
    gbstep    INTEGER
            the offset of info, internal use

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_iamax_batched
*******************************************************************************/
extern "C" magma_int_t
magma_izamax_batched(magma_int_t length,
                     magmaDoubleComplex **x_array, magma_int_t xi, magma_int_t xj, magma_int_t incx,
                     magma_int_t step,  magma_int_t lda,
                     magma_int_t** ipiv_array, magma_int_t *info_array,
                     magma_int_t gbstep, magma_int_t batchCount, magma_queue_t queue)
{
    if (length == 0 ) return 0;

    dim3 grid(batchCount, 1, 1);
    dim3 threads(zamax, 1, 1);

    int chunk = magma_ceildiv( length, zamax );
    izamax_kernel_batched<<< grid, threads, zamax * (sizeof(double) + sizeof(int)), queue->hip_stream() >>>
        (length, chunk, x_array, xi, xj, incx, step, lda, ipiv_array, info_array, gbstep);

    return 0;
}


/******************************************************************************/
// For use in magma_izamax_native only
// hipblasIzamax always writes 32bit pivots, so make sure it is magma_int_t
__global__ void magma_zpivcast(magma_int_t* dipiv)
{
    // uses only 1 thread
    int* address = (int*)dipiv;
    int pivot = *address;          // read the value written by cuBLAS (int)
    *dipiv = (magma_int_t)pivot;    // write it back in the same address as dipiv
}

/******************************************************************************/
extern "C" magma_int_t
magma_izamax_native( magma_int_t length,
                     magmaDoubleComplex_ptr x, magma_int_t incx,
                     magma_int_t step,  magma_int_t lda,
                     magma_int_t* ipiv, magma_int_t *info,
                     magma_int_t gbstep, magma_queue_t queue)
{
    if (length == 0 ) return 0;

    // TODO: decide the best izamax for all precisions
    if( length <= 15360 ) {
        dim3 grid(1, 1, 1);
        dim3 threads(zamax, 1, 1);

        int chunk = magma_ceildiv( length, zamax );
        izamax_kernel_native<<< grid, threads, zamax * (sizeof(double) + sizeof(int)), queue->hip_stream() >>>
            (length, chunk, x, incx, step, lda, ipiv, info, gbstep);
    }
    else {
        hipblasPointerMode_t ptr_mode;
        hipblasGetPointerMode(queue->cublas_handle(), &ptr_mode);
        hipblasSetPointerMode(queue->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE);

        hipblasIzamax(queue->cublas_handle(), length, x + step * lda + step, 1, (int*)(ipiv+step));
        magma_zpivcast<<< 1, 1, 0, queue->hip_stream() >>>( ipiv+step );

        hipblasSetPointerMode(queue->cublas_handle(), ptr_mode);
        adjust_ipiv( ipiv+step, 1, step, queue);
    }
    return 0;
}


/******************************************************************************/
__device__
void zswap_device( magma_int_t n,
                   magmaDoubleComplex_ptr x, magma_int_t incx,
                   magma_int_t step, magma_int_t* ipiv)
{
    const int tx = threadIdx.x;

    __shared__ int jp;

    if (tx == 0){
        jp = ipiv[step] - 1;
    }
    __syncthreads();

    if (jp == step) return; // no pivot

    if (tx < n) {
        magmaDoubleComplex tmp = x[jp + tx * incx];
        x[jp + tx * incx] = x[step + tx * incx];
        x[step + tx * incx] = tmp;
    }
}


/******************************************************************************/
__global__
void zswap_kernel_batched(
        magma_int_t n,
        magmaDoubleComplex **x_array, magma_int_t xi, magma_int_t xj, magma_int_t incx,
        magma_int_t step, magma_int_t** ipiv_array)
{
    const int batchid = blockIdx.x;
    magmaDoubleComplex *x = x_array[batchid] + xj * incx + xi;
    magma_int_t *ipiv = ipiv_array[batchid] + xi;

    zswap_device(n, x, incx, step, ipiv);
}


/******************************************************************************/
__global__
void zswap_kernel_native( magma_int_t n,
                          magmaDoubleComplex_ptr x, magma_int_t incx,
                          magma_int_t step, magma_int_t* ipiv)
{
    zswap_device(n, x, incx, step, ipiv);
}


/***************************************************************************//**
    Purpose
    -------

    zswap two row in x.  index (ipiv[step]-1)-th and index step -th

    This is an internal routine that might have many assumption.

    Arguments
    ---------

    @param[in]
    n       INTEGER
            On entry, n specifies the size of vector x. n >= 0.


    @param[in]
    dA_array  Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array of dimension


    @param[in]
    ai      INTEGER
            Row offset, internal use.

    @param[in]
    aj      INTEGER
            Column offset, internal use.

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    step    INTEGER
            The starting address of matrix C in A.  LDDA >= max(1,M).

    @param[out]
    ipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).


    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_swap_batched
*******************************************************************************/
extern "C" magma_int_t
magma_zswap_batched( magma_int_t n,
                     magmaDoubleComplex **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t incx,
                     magma_int_t step, magma_int_t** ipiv_array,
                     magma_int_t batchCount, magma_queue_t queue)
{
    /*
    zswap two row: (ipiv[step]-1)th and step th
    */
    if ( n  > MAX_NTHREADS)
    {
        fprintf( stderr, "%s nb=%lld > %lld, not supported\n",
                 __func__, (long long) n, (long long) MAX_NTHREADS );
        return -15;
    }
    dim3 grid(batchCount, 1, 1);
    dim3 threads(zamax, 1, 1);

    zswap_kernel_batched
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, dA_array, ai, aj, incx, step, ipiv_array);
    return 0;
}


/******************************************************************************/
extern "C" void
magma_zswap_native( magma_int_t n, magmaDoubleComplex_ptr x, magma_int_t incx,
                    magma_int_t step, magma_int_t* ipiv,
                    magma_queue_t queue)
{
    /*
    zswap two row: (ipiv[step]-1)th and step th
    */
    if ( n  > MAX_NTHREADS){
        fprintf( stderr, "%s nb=%lld > %lld, not supported\n",
                 __func__, (long long) n, (long long) MAX_NTHREADS );
    }
    dim3 grid(1, 1, 1);
    dim3 threads(zamax, 1, 1);

    zswap_kernel_native
        <<< grid, threads, 0, queue->hip_stream() >>>
        (n, x, incx, step, ipiv);
}


/******************************************************************************/
template<int N>
__device__
void zscal_zgeru_device( int m, int step,
                         magmaDoubleComplex_ptr dA, int lda,
                         magma_int_t *info, int gbstep)
{
    const int tx  = threadIdx.x;
    const int gtx = blockIdx.x * blockDim.x + tx;
    // checkinfo to avoid computation of the singular matrix
    if( (*info) != 0 ) return;

    magmaDoubleComplex_ptr A = dA + step + step * lda;
    magmaDoubleComplex rA[N], reg;
    __shared__ magmaDoubleComplex shared_y[N];

    if (tx < N) {
        shared_y[tx] = A[lda * tx];
    }
    __syncthreads();

    if (shared_y[0] == MAGMA_Z_ZERO) {
        (*info) = step + gbstep + 1;
        return;
    }

    // terminate threads that are out of the range
    if (gtx == 0 || gtx >= m) return;

    reg = MAGMA_Z_DIV(MAGMA_Z_ONE, shared_y[0]);
    #pragma unroll
    for(int i = 0; i < N; i++)
        rA[i] = A[ i* lda + gtx ];

    rA[0] *= reg;

    #pragma unroll
    for(int i = 1; i < N; i++)
        rA[i] -= rA[0] * shared_y[i];

    #pragma unroll
    for(int i = 0; i < N; i++)
        A[gtx + i * lda] = rA[i];
}


/******************************************************************************/
__device__
void zscal_zgeru_generic_device( int m, int n, int step,
                         magmaDoubleComplex_ptr dA, int lda,
                         magma_int_t *info, int gbstep)
{
    const int tx  = threadIdx.x;
    const int gtx = blockIdx.x * blockDim.x + tx;
    // checkinfo to avoid computation of the singular matrix
    if( (*info) != 0 ) return;
    if (gtx == 0 || gtx >= m) return;

    magmaDoubleComplex_ptr A = dA + step + step * lda;
    magmaDoubleComplex rA, reg;

    if (A[0] == MAGMA_Z_ZERO) {
        (*info) = step + gbstep + 1;
        return;
    }

    reg = MAGMA_Z_DIV(MAGMA_Z_ONE, A[0]);
    rA  = A[ gtx ];
    rA *= reg;

    A[ gtx ] = rA;
    #pragma unroll
    for(int i = 1; i < n; i++)
        A[i * lda + gtx] -= rA * A[i * lda + 0];

}


/******************************************************************************/
template<int N>
__global__
void zscal_zgeru_1d_kernel_native( int m, int step,
                                magmaDoubleComplex_ptr dA, int lda,
                                magma_int_t *info, int gbstep)
{
    // This dev function has a return statement inside, be sure
    // not to merge it with another dev function. Otherwise, the
    // return statement should be converted into an if-statement
    zscal_zgeru_device<N>(m, step, dA, lda, info, gbstep);
}


/******************************************************************************/
__global__
void zscal_zgeru_1d_generic_kernel_native( int m, int n, int step,
                                magmaDoubleComplex_ptr dA, int lda,
                                magma_int_t *info, int gbstep)
{
    // This dev function has a return statement inside, be sure
    // not to merge it with another dev function. Otherwise, the
    // return statement should be converted into an if-statement
    zscal_zgeru_generic_device(m, n, step, dA, lda, info, gbstep);
}


/******************************************************************************/
template<int N>
__global__
void zscal_zgeru_1d_kernel_batched(int m, int step, magmaDoubleComplex **dA_array, int ai, int aj, int lda, magma_int_t *info_array, int gbstep)
{
    const int batchid = blockIdx.z;
    magmaDoubleComplex* dA = dA_array[batchid] + aj * lda + ai;
    magma_int_t *info = &info_array[batchid];
    zscal_zgeru_device<N>(m, step, dA, lda, info, gbstep);
}


/******************************************************************************/
__global__
void zscal_zgeru_1d_generic_kernel_batched(int m, int n, int step, magmaDoubleComplex **dA_array, int ai, int aj, int lda, magma_int_t *info_array, int gbstep)
{
    const int batchid = blockIdx.z;
    magmaDoubleComplex* dA = dA_array[batchid] + aj * lda + ai;
    magma_int_t *info = &info_array[batchid];
    zscal_zgeru_generic_device(m, n, step, dA, lda, info, gbstep);
}


/******************************************************************************/
extern "C"
magma_int_t magma_zscal_zgeru_batched(magma_int_t m, magma_int_t n, magma_int_t step,
                                      magmaDoubleComplex **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t lda,
                                      magma_int_t *info_array, magma_int_t gbstep,
                                      magma_int_t batchCount, magma_queue_t queue)
{
    /*
    Specialized kernel which merged zscal and zgeru the two kernels
    1) zscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a zgeru Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    if ( n == 0) return 0;
    if ( n > MAX_NTHREADS ) {
        fprintf( stderr, "%s nb=%lld, > %lld, not supported\n", __func__, (long long) n, (long long) MAX_NTHREADS );
        return -15;
    }

    magma_int_t max_batchCount = queue->get_maxBatch();
    const int tbx = MAX_NTHREADS / 2;
    dim3 threads(tbx, 1, 1);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid(magma_ceildiv(m,tbx), 1, ibatch);

        switch(n){
            case  1:zscal_zgeru_1d_kernel_batched< 1><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  2:zscal_zgeru_1d_kernel_batched< 2><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  3:zscal_zgeru_1d_kernel_batched< 3><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  4:zscal_zgeru_1d_kernel_batched< 4><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  5:zscal_zgeru_1d_kernel_batched< 5><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  6:zscal_zgeru_1d_kernel_batched< 6><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  7:zscal_zgeru_1d_kernel_batched< 7><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            case  8:zscal_zgeru_1d_kernel_batched< 8><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);break;
            default:zscal_zgeru_1d_generic_kernel_batched<<<grid, threads, 0, queue->hip_stream()>>>(m, n, step, dA_array+i, ai, aj, lda, info_array+i, gbstep);
        }
    }
    return 0;
}


/******************************************************************************/
extern "C"
magma_int_t
magma_zscal_zgeru_native(
    magma_int_t m, magma_int_t n, magma_int_t step,
    magmaDoubleComplex_ptr dA, magma_int_t lda,
    magma_int_t *info, magma_int_t gbstep,
    magma_queue_t queue)
{
    /*
    Specialized kernel which merged zscal and zgeru the two kernels
    1) zscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a zgeru Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    if ( n == 0) return 0;
    if ( n > MAX_NTHREADS ) {
        fprintf( stderr, "%s nb=%lld, > %lld, not supported\n", __func__, (long long) n, (long long) MAX_NTHREADS );
        return -15;
    }
    const int tbx = MAX_NTHREADS / 2;
    dim3 grid(magma_ceildiv(m,tbx), 1, 1);
    dim3 threads(tbx, 1, 1);
    switch(n){
        case 1:zscal_zgeru_1d_kernel_native<1><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 2:zscal_zgeru_1d_kernel_native<2><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 3:zscal_zgeru_1d_kernel_native<3><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 4:zscal_zgeru_1d_kernel_native<4><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 5:zscal_zgeru_1d_kernel_native<5><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 6:zscal_zgeru_1d_kernel_native<6><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 7:zscal_zgeru_1d_kernel_native<7><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        case 8:zscal_zgeru_1d_kernel_native<8><<<grid, threads, 0, queue->hip_stream()>>>( m, step, dA, lda, info, gbstep);break;
        default:zscal_zgeru_1d_generic_kernel_native<<<grid, threads, 0, queue->hip_stream()>>>( m, n, step, dA, lda, info, gbstep);
    }
    return 0;
}


/******************************************************************************/
__global__
void zgetf2trsm_kernel_batched(int ib, int n, magmaDoubleComplex **dA_array, int step, int lda)
{
    /*
        this kernel does the safe nonblocked TRSM operation
        B = A^-1 * B
    */
    const int batchid = blockIdx.x;

    magmaDoubleComplex *A_start = dA_array[batchid];
    magmaDoubleComplex *A = &(A_start[step + step * lda]);
    magmaDoubleComplex *B = &(A_start[step + (step+ib) * lda]);
    magmaDoubleComplex *shared_a = shared_data;
    magmaDoubleComplex *shared_b = shared_data+ib*ib;

    int tid = threadIdx.x;
    int i,d;


    // Read A and B at the same time to the shared memory (shared_a shared_b)
    // note that shared_b = shared_a+ib*ib so its contiguous
    // I can make it in one loop reading
    if ( tid < ib) {
        #pragma unroll
        for (i=0; i < n+ib; i++) {
            shared_a[tid + i*ib] = A[tid + i*lda];
        }
    }
    __syncthreads();

    if (tid < n) {
        #pragma unroll
        for (d=0;  d < ib-1; d++) {
            for (i=d+1; i < ib; i++) {
                shared_b[i+tid*ib] += (MAGMA_Z_NEG_ONE) * shared_a[i+d*ib] * shared_b[d+tid*ib];
            }
        }
    }
    __syncthreads();

    // write back B
    if ( tid < ib) {
        #pragma unroll
        for (i=0; i < n; i++) {
            B[tid + i*lda] = shared_b[tid + i*ib];
        }
    }
}


/***************************************************************************//**
    Purpose
    -------

    zgetf2trsm solves one of the matrix equations on gpu

     B = C^-1 * B

    where C, B are part of the matrix A in dA_array,

    This version load C, B into shared memory and solve it
    and copy back to GPU device memory.
    This is an internal routine that might have many assumption.

    Arguments
    ---------
    @param[in]
    ib       INTEGER
            The number of rows/columns of each matrix C, and rows of B.  ib >= 0.

    @param[in]
    n       INTEGER
            The number of columns of each matrix B.  n >= 0.

    @param[in,out]
    dA_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array on the GPU, dimension (LDDA,N).
            On entry, each pointer is an M-by-N matrix to be factored.
            On exit, the factors L and U from the factorization
            A = P*L*U; the unit diagonal elements of L are not stored.

    @param[in]
    ldda    INTEGER
            The leading dimension of each array A.  LDDA >= max(1,M).

    @param[in]
    step    INTEGER
            The starting address of matrix C in A.  LDDA >= max(1,M).

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_getf2_batched
*******************************************************************************/
extern "C" void
magma_zgetf2trsm_batched(magma_int_t ib, magma_int_t n, magmaDoubleComplex **dA_array,
                         magma_int_t step, magma_int_t ldda,
                         magma_int_t batchCount, magma_queue_t queue)
{
    if ( n == 0 || ib == 0 ) return;
    size_t shared_size = sizeof(magmaDoubleComplex)*(ib*(ib+n));

    // TODO TODO TODO
    if ( shared_size > (MAX_SHARED_ALLOWED*1024) ) // limit the shared memory to 46K leaving 2K for extra
    {
        fprintf( stderr, "%s: error out of shared memory\n", __func__ );
        return;
    }

    dim3 grid(batchCount, 1, 1);
    dim3 threads(max(n,ib), 1, 1);

    zgetf2trsm_kernel_batched
    <<< grid, threads, shared_size, queue->hip_stream() >>>
    (ib, n, dA_array, step, ldda);
}


/******************************************************************************/
template<int NB>
__global__ void
zgetf2trsm_2d_kernel( int m, int n,
                           magmaDoubleComplex_ptr dA, int ldda,
                           magmaDoubleComplex_ptr dB, int lddb)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ magmaDoubleComplex sA[NB * NB];
    __shared__ magmaDoubleComplex sB[NB * NB];

    // init sA & sB
    sA[ ty * NB + tx ] = MAGMA_Z_ZERO;
    sB[ ty * NB + tx ] = MAGMA_Z_ZERO;

    const int nblocks = magma_ceildiv(n, NB);
    const int n_ = n - (nblocks-1) * NB;

    // load A
    if( ty < m && tx < m && tx > ty){
        sA[ty * NB + tx] = dA[ty * ldda + tx];
    }

    if( ty == tx ){
        // ignore diagonal elements
        sA[tx * NB + tx] = MAGMA_Z_ONE;
    }
    __syncthreads();

    #pragma  unroll
    for(int s = 0; s < nblocks-1; s++){
        // load B
        if( tx < m ){
            sB[ ty * NB + tx] = dB[ ty * lddb + tx ];
        }

        // no need to sync because each thread column is less than 32
        // solve
        #pragma unroll
        for(int i = 0; i < NB; i++){
            if(tx >  i){
                 sB[ ty * NB + tx ] -= sA[ i * NB + tx ] * sB[ ty * NB + i ];
            }
        }

        // write B
        if( tx < m){
            dB[ ty * lddb + tx ] = sB[ ty * NB + tx ];
        }
        dB += NB * lddb;
    }

    // last, possible partial, block
    if( ty < n_ && tx < m){
        sB[ ty * NB + tx] = dB[ ty * lddb + tx ];
    }

    #pragma unroll
    for(int i = 0; i < NB; i++){
        if(tx >  i){
             sB[ ty * NB + tx ] -= sA[ i * NB + tx ] * sB[ ty * NB + i ];
        }
    }

    if( ty < n_ && tx < m){
        dB[ ty * lddb + tx ] = sB[ ty * NB + tx ];
    }
}


/******************************************************************************/
extern"C" void
magma_zgetf2trsm_2d_native(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_ptr dB, magma_int_t lddb,
    magma_queue_t queue)
{
    if( m > 32 ){
        magma_ztrsm( MagmaLeft, MagmaLower, MagmaNoTrans, MagmaUnit,
                     m, n, MAGMA_Z_ONE,
                     dA, ldda,
                     dB, lddb, queue );
        return;
    }

    const int m8 = magma_roundup(m, 8);
    dim3 grid(1, 1, 1);
    dim3 threads(m8, m8, 1);

    switch(m8){
        case  8:zgetf2trsm_2d_kernel< 8><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        case 16:zgetf2trsm_2d_kernel<16><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        case 24:zgetf2trsm_2d_kernel<24><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        case 32:zgetf2trsm_2d_kernel<32><<<grid, threads, 0, queue->hip_stream() >>>( m, n, dA, ldda, dB, lddb ); break;
        default:;
    }
}


/******************************************************************************/
static __device__ void
zupdate_device(int m, int step, magmaDoubleComplex* x, int ldx,  magmaDoubleComplex *A, int lda)
{
    int tid = threadIdx.x;
    int nchunk = magma_ceildiv( m, MAX_NTHREADS );
    int indx;
    //magmaDoubleComplex reg = MAGMA_Z_ZERO;

    // update the current column by all the previous one
    #pragma unroll
    for (int i=0; i < step; i++) {
        for (int s=0; s < nchunk; s++)
        {
            indx = tid + s * MAX_NTHREADS;
            if ( indx > i  && indx < m ) {
                A[indx] -=  A[i] * x[indx + i*ldx];
                //printf("         @ step %d tid %d updating x[tid]*y[i]=A %5.3f %5.3f = %5.3f  at i %d\n", step, tid, x[tid + i*ldx], A[i], A[tid],i);
            }
        }
        __syncthreads();
    }

    //printf("         @ step %d tid %d adding %5.3f to A %5.3f make it %5.3f\n",step,tid,-reg,A[tid],A[tid]-reg);
}


/******************************************************************************/
static __device__ void
zscal5_device(int m, magmaDoubleComplex* x, magmaDoubleComplex alpha)
{
    int tid = threadIdx.x;
    int nchunk = magma_ceildiv( m, MAX_NTHREADS );

    for (int s=0; s < nchunk; s++)
    {
        if ( (tid + s * MAX_NTHREADS) < m ) {
            #if 0
            x[tid + s * MAX_NTHREADS] *= MAGMA_Z_DIV(MAGMA_Z_ONE, alpha);
            #else
            x[tid + s * MAX_NTHREADS] = x[tid + s * MAX_NTHREADS]/alpha;
            #endif
        }
    }
    __syncthreads();
}


/******************************************************************************/
__global__ void
zcomputecolumn_kernel_shared_batched( int m, int paneloffset, int step,
                                      magmaDoubleComplex **dA_array, int ai, int aj,
                                      int lda, magma_int_t **ipiv_array, magma_int_t *info_array, int gbstep)
{
    const int batchid = blockIdx.x;
    int gboff = paneloffset+step;
    magma_int_t *ipiv           = ipiv_array[batchid] + ai;
    magmaDoubleComplex *A_start = dA_array[batchid] + aj * lda + ai;
    magmaDoubleComplex *A0j     = &(A_start[paneloffset + (paneloffset+step) * lda]);
    magmaDoubleComplex *A00     = &(A_start[paneloffset + paneloffset * lda]);

    magmaDoubleComplex *shared_A = shared_data;
    __shared__ double  shared_x[zamax];
    __shared__ int     shared_idx[zamax];
    __shared__ magmaDoubleComplex alpha;
    int tid = threadIdx.x;

    // checkinfo to avoid computation of the singular matrix
    if (info_array[batchid] != 0 ) return;


    int nchunk = magma_ceildiv( m, MAX_NTHREADS );
    // read the current column from dev to shared memory
    for (int s=0; s < nchunk; s++)
    {
        if ( (tid + s * MAX_NTHREADS) < m ) shared_A[tid + s * MAX_NTHREADS] = A0j[tid + s * MAX_NTHREADS];
    }
    __syncthreads();

    // update this column
    if ( step > 0 ) {
        zupdate_device( m, step, A00, lda, shared_A, 1);
        __syncthreads();
    }

    // if ( tid < (m-step) ) // DO NO TPUT THE IF CONDITION HERE SINCE izamax_devfunc HAS __syncthreads INSIDE.
    // So let all htreads call this routine it will handle correctly based on the size
    // note that izamax need only 128 threads, s
    izamax_devfunc(m-step, shared_A+step, 1, shared_x, shared_idx);
    if (tid == 0) {
        ipiv[gboff]  = shared_idx[0] + gboff + 1; // Fortran Indexing
        alpha = shared_A[shared_idx[0]+step];
        //printf("@ step %d ipiv=%d where gboff=%d  shared_idx %d alpha %5.3f\n",step,ipiv[gboff],gboff,shared_idx[0],alpha);
        if (shared_x[0] == MAGMA_D_ZERO) {
            info_array[batchid] = shared_idx[0] + gboff + gbstep + 1;
        }
    }
    __syncthreads();
    if (shared_x[0] == MAGMA_D_ZERO) return;
    __syncthreads();

    // DO NO PUT THE IF CONDITION HERE SINCE izamax_devfunc HAS __syncthreads INSIDE.
    zscal5_device( m-step, shared_A+step, alpha);

    // put back the pivot that has been scaled with itself menaing =1
    if (tid == 0)  shared_A[shared_idx[0] + step] = alpha;
    __syncthreads();

    // write back from shared to dev memory
    for (int s=0; s < nchunk; s++)
    {
        if ( (tid + s * MAX_NTHREADS) < m )
        {
            A0j[tid + s * MAX_NTHREADS] = shared_A[tid + s * MAX_NTHREADS];
            //printf("@ step %d tid %d updating A=x*alpha after A= %5.3f\n",step,tid,shared_A[tid]);
        }
    }
    __syncthreads();
}


/******************************************************************************/
extern "C"
magma_int_t magma_zcomputecolumn_batched( magma_int_t m, magma_int_t paneloffset, magma_int_t step,
                                          magmaDoubleComplex **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t lda,
                                          magma_int_t **ipiv_array,
                                          magma_int_t *info_array, magma_int_t gbstep,
                                          magma_int_t batchCount, magma_queue_t queue)
{
    /*
    Specialized kernel which merged zscal and zgeru the two kernels
    1) zscale the first column vector A(1:M-1,0) with 1/A(0,0);
    2) Performe a zgeru Operation for trailing matrix of A(1:M-1,1:N-1) += alpha*x*y**T, where
       alpha := -1.0; x := A(1:M-1,0) and y:= A(0,1:N-1);
    */
    if ( m == 0) return 0;

    size_t all_shmem_size = zamax*(sizeof(double)+sizeof(int)) + (m+2)*sizeof(magmaDoubleComplex);
    if ( all_shmem_size >  (MAX_SHARED_ALLOWED*1024) ) // limit the shared memory to 44K leaving 4K for extra
    {
        fprintf( stderr, "%s error out of shared memory\n", __func__ );
        return -20;
    }

    size_t shared_size = sizeof(magmaDoubleComplex)*m;
    dim3 grid(batchCount, 1, 1);
    dim3 threads(min(m, MAX_NTHREADS), 1, 1);

    zcomputecolumn_kernel_shared_batched
    <<< grid, threads, shared_size, queue->hip_stream() >>>
    (m, paneloffset, step, dA_array, ai, aj, lda, ipiv_array, info_array, gbstep);

    return 0;
}


/******************************************************************************/
template<int WIDTH>
__device__ void
zgetf2_fused_device( int m, magmaDoubleComplex* dA, int ldda, magma_int_t* dipiv,
                   magmaDoubleComplex* swork, magma_int_t *info, int gbstep)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    magmaDoubleComplex rA[WIDTH] = {MAGMA_Z_ZERO};
    magmaDoubleComplex reg       = MAGMA_Z_ZERO;
    magmaDoubleComplex update    = MAGMA_Z_ZERO;

    int max_id, rowid = tx;
    int linfo = (gbstep == 0) ? 0 : *info;
    double rx_abs_max = MAGMA_D_ZERO;
    // check from previous calls if the panel factorization failed previously
    // this is necessary to report the correct info value
    //if(gbstep > 0 && *info != 0) return;

    magmaDoubleComplex *sx = (magmaDoubleComplex*)(swork);
    double* dsx = (double*)(sx + blockDim.y * WIDTH);
    int* isx    = (int*)(dsx + blockDim.y * m);
    int* sipiv  = (int*)(isx + blockDim.y * m);
    sx    += ty * WIDTH;
    dsx   += ty * m;
    isx   += ty * m;
    sipiv += ty * WIDTH;

    // init sipiv
    if(tx < WIDTH){
        sipiv[tx] = 0;
    }

    // read
    #pragma unroll
    for(int i = 0; i < WIDTH; i++){
        rA[i] = dA[ i * ldda + tx ];
    }

    #pragma unroll
    for(int i = 0; i < WIDTH; i++){
        // izamax and find pivot
        dsx[ rowid ] = fabs(MAGMA_Z_REAL( rA[i] )) + fabs(MAGMA_Z_IMAG( rA[i] ));
        isx[ tx ] = tx;
        __syncthreads();
        magma_getidmax_n(m-i, tx, dsx+i, isx+i); // this devfunc has syncthreads at the end
        rx_abs_max = dsx[i];
        max_id = isx[i];
        linfo  = ( rx_abs_max == MAGMA_D_ZERO && linfo == 0) ? (gbstep+i+1) : linfo;
        update = ( rx_abs_max == MAGMA_D_ZERO ) ? MAGMA_Z_ZERO : MAGMA_Z_ONE;
        __syncthreads();

        if(rowid == max_id){
            sipiv[i] = max_id;
            rowid = i;
            #pragma unroll
            for(int j = 0; j < WIDTH; j++){
                sx[j] = update * rA[j];
            }
        }
        else if(rowid == i){
            rowid = max_id;
        }
        __syncthreads();

        reg = (linfo == 0 ) ? MAGMA_Z_DIV(MAGMA_Z_ONE, sx[i] ) : MAGMA_Z_ONE;
        // scal and ger
        if( rowid > i ){
            rA[i] *= reg;
            #pragma unroll
            for(int j = i+1; j < WIDTH; j++){
                rA[j] -= rA[i] * sx[j];
            }
        }
    }

    if(tx == 0){
        (*info) = (magma_int_t)( linfo );
    }
    // write
    if(tx < WIDTH){
        dipiv[tx] = (magma_int_t)(sipiv[tx] + 1); // fortran indexing
        //printf("--- ipiv[%d] --- = %d\n", tx, dipiv[tx]);
    }

    #pragma unroll
    for(int i = 0; i < WIDTH; i++){
        dA[ i * ldda + rowid ] = rA[i];
    }
}

/******************************************************************************/
extern __shared__ magmaDoubleComplex zdata[];
template<int WIDTH>
__global__ void
zgetf2_fused_batched_kernel( int m,
                           magmaDoubleComplex** dA_array, int ai, int aj, int ldda,
                           magma_int_t** dipiv_array, magma_int_t* info_array, int batchCount)
{
     magmaDoubleComplex* swork = (magmaDoubleComplex*)zdata;
     const int batchid = blockIdx.x * blockDim.y + threadIdx.y;
     if(batchid >= batchCount)return;
     zgetf2_fused_device<WIDTH>(
             m, dA_array[batchid] + aj * ldda + ai, ldda,
             dipiv_array[batchid] + ai,
             swork, &info_array[batchid], aj);
}


/***************************************************************************//**
    Purpose
    -------
    magma_zgetf2_reg_batched computes an LU factorization of a general M-by-N matrix A
    using partial pivoting with row interchanges. This routine is used for batch LU panel
    factorization, and has specific assumption about the value of N

    The factorization has the form
        A = P * L * U
    where P is a permutation matrix, L is lower triangular with unit
    diagonal elements (lower trapezoidal if m > n), and U is upper
    triangular (upper trapezoidal if m < n).

    This is a right-looking unblocked version of the algorithm. The routine is a batched
    version that factors batchCount M-by-N matrices in parallel.

    This version load an entire matrix (m*n) into registers and factorize it with pivoting
    and copy back to GPU device memory.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of each matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of each matrix A.  ib >= 0.

    @param[in,out]
    dA_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array on the GPU, dimension (LDDA,N).
            On entry, each pointer is an M-by-N matrix to be factored.
            On exit, the factors L and U from the factorization
            A = P*L*U; the unit diagonal elements of L are not stored.

    @param[in]
    ai      INTEGER
            Row offset for A.

    @param[in]
    aj      INTEGER
            Column offset for A.

    @param[in]
    ldda    INTEGER
            The leading dimension of each array A.  LDDA >= max(1,M).

    @param[out]
    dipiv_array  Array of pointers, dimension (batchCount), for corresponding matrices.
            Each is an INTEGER array, dimension (min(M,N))
            The pivot indices; for 1 <= i <= min(M,N), row i of the
            matrix was interchanged with row IPIV(i).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
                  or another error occured, such as memory allocation failed.
      -     > 0:  if INFO = i, U(i,i) is exactly zero. The factorization
                  has been completed, but the factor U is exactly
                  singular, and division by zero will occur if it is used
                  to solve a system of equations.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_getf2_batched
*******************************************************************************/
extern "C" magma_int_t
magma_zgetf2_fused_batched(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex **dA_array, magma_int_t ai, magma_int_t aj, magma_int_t ldda,
    magma_int_t **dipiv_array,
    magma_int_t *info_array, magma_int_t batchCount,
    magma_queue_t queue)
{
    if(m < 0 || m > ZGETF2_FUSED_BATCHED_MAX_ROWS) {
        fprintf( stderr, "%s: m = %4lld not supported, must be between 0 and %4lld\n",
                 __func__, (long long) m, (long long) ZGETF2_FUSED_BATCHED_MAX_ROWS);
        return -1;
    }
    else if(n < 0 || n > 32){
        fprintf( stderr, "%s: n = %4lld not supported, must be between 0 and %4lld\n",
                 __func__, (long long) m, (long long) 32);
        return -2;
    }
    magma_int_t ntcol = (m > 32)? 1 : (2 * (32/m));

    magma_int_t shared_size = 0;
    shared_size += n * sizeof(magmaDoubleComplex);
    shared_size += m * sizeof(double);
    shared_size += m * sizeof(int);    // not magma_int_t
    shared_size += n * sizeof(int);    // not magma_int_t
    shared_size *= ntcol;

    dim3 grid(magma_ceildiv(batchCount,ntcol), 1, 1);
    dim3 threads(m, ntcol, 1);

    switch(n)
    {
        case  1: zgetf2_fused_batched_kernel< 1><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  2: zgetf2_fused_batched_kernel< 2><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  3: zgetf2_fused_batched_kernel< 3><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  4: zgetf2_fused_batched_kernel< 4><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  5: zgetf2_fused_batched_kernel< 5><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  6: zgetf2_fused_batched_kernel< 6><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  7: zgetf2_fused_batched_kernel< 7><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  8: zgetf2_fused_batched_kernel< 8><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case  9: zgetf2_fused_batched_kernel< 9><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 10: zgetf2_fused_batched_kernel<10><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 11: zgetf2_fused_batched_kernel<11><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 12: zgetf2_fused_batched_kernel<12><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 13: zgetf2_fused_batched_kernel<13><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 14: zgetf2_fused_batched_kernel<14><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 15: zgetf2_fused_batched_kernel<15><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 16: zgetf2_fused_batched_kernel<16><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 17: zgetf2_fused_batched_kernel<17><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 18: zgetf2_fused_batched_kernel<18><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 19: zgetf2_fused_batched_kernel<19><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 20: zgetf2_fused_batched_kernel<20><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 21: zgetf2_fused_batched_kernel<21><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 22: zgetf2_fused_batched_kernel<22><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 23: zgetf2_fused_batched_kernel<23><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 24: zgetf2_fused_batched_kernel<24><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 25: zgetf2_fused_batched_kernel<25><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 26: zgetf2_fused_batched_kernel<26><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 27: zgetf2_fused_batched_kernel<27><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 28: zgetf2_fused_batched_kernel<28><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 29: zgetf2_fused_batched_kernel<29><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 30: zgetf2_fused_batched_kernel<30><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 31: zgetf2_fused_batched_kernel<31><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        case 32: zgetf2_fused_batched_kernel<32><<<grid, threads, shared_size, queue->hip_stream()>>>(m, dA_array, ai, aj, ldda, dipiv_array, info_array, batchCount); break;
        default: fprintf( stderr, "%s: n = %4lld is not supported \n", __func__, (long long) n);
    }
    return 0;
}