#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from magmablas/ztrtri_lower.cu, normal z -> s, Thu Oct  8 23:05:35 2020

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       This file implements lower case, and is called by strtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "strtri.cuh"
#include "strtri_lower_device.cuh"


/******************************************************************************/
__global__ void
strtri_diag_lower_kernel(
    magma_diag_t diag, int n, const float *A, int lda, float *d_dinvA)
{
    strtri_diag_lower_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_sgemm16_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm16_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm16_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm16_part2_lower_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm32_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm32_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm32_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm32_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm64_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm64_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm_above64_part1_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm_above64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm_above64_part2_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm_above64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_sgemm_above64_part3_lower_kernel(
    int n, const float *Ain, int lda, float *d_dinvA, int jb, int npages)
{
    triple_sgemm_above64_part3_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}
