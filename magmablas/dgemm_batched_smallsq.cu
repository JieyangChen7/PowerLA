#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Ahmad Abdelfattah
       @author Azzam Haidar

       @generated from magmablas/zgemm_batched_smallsq.cu, normal z -> d, Thu Oct  8 23:05:36 2020
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

#define SLDA(N)    ( (N==15||N==23||N==31)? N : (N+1) )

extern __shared__ double zdata[];
template<int N>
__global__ void
dgemm_batched_smallsq_kernel(
        const magma_trans_t transA, magma_trans_t transB, 
        const double alpha, double const * const * dA_array, int ai, int aj, int ldda, 
                                        double const * const * dB_array, int bi, int bj, int lddb, 
        const double beta,  double**               dC_array, int ci, int cj, int lddc, 
        const int batchCount)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    const int bx = blockIdx.x;
    
    const int batchid = bx * blockDim.z + tz;
    if(batchid >= batchCount) return;
    
    const double* __restrict__ dA = dA_array[batchid] + aj * ldda + ai;
    const double* __restrict__ dB = dB_array[batchid] + bj * lddb + bi;
          double* __restrict__ dC = dC_array[batchid] + cj * lddc + ci;
    
    double rC = MAGMA_D_ZERO; 
    double rTmp = MAGMA_D_ZERO; 
    
    const int slda = SLDA(N);
    const int sldb = SLDA(N);
    double* sA = (double*)(zdata);
    double* sB = (double*)(zdata + blockDim.z * slda * N);
    
    sA += tz * slda * N;
    sB += tz * sldb * N;
    
    // read A & B 
    if(transA == MagmaNoTrans){
        sA[ty * slda + tx] = dA[ty * ldda + tx];
    }
    else{
        sA[tx * slda + ty] = (transA == MagmaTrans) ? dA[ty * ldda + tx] : MAGMA_D_CONJ( dA[ty * ldda + tx] );
    }

    if(transB == MagmaNoTrans){
        sB[ty * sldb + tx] = dB[ty * lddb + tx];
    }
    else{
        sB[tx * sldb + ty] = (transB == MagmaTrans) ? dB[ty * lddb + tx] : MAGMA_D_CONJ( dB[ty * lddb + tx] );
    }
    __syncthreads(); 

    if(beta != MAGMA_D_ZERO){
        rC = beta * dC[ty * lddc + tx];
    }

    // multiply
    rTmp = MAGMA_D_ZERO;
    #pragma unroll
    for(int j = 0; j < N; j++){
        rTmp += sA[j * slda + tx] * sB[ty * sldb + j]; 
    }
    rC += alpha * rTmp;

    // write from rC
    dC[ty * lddc + tx] = rC;
}


extern "C" void 
magmablas_dgemm_batched_smallsq(
    magma_trans_t transA, magma_trans_t transB, 
    magma_int_t m, magma_int_t n, magma_int_t k, 
    double alpha,
    double const * const * dA_array, magma_int_t ai, magma_int_t aj, magma_int_t ldda, 
    double const * const * dB_array, magma_int_t bi, magma_int_t bj, magma_int_t lddb,
    double beta,
    double **dC_array, magma_int_t ci, magma_int_t cj, magma_int_t lddc, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    if      ( transA != MagmaNoTrans && transA != MagmaTrans && transA != MagmaConjTrans )
        info = -1;
    else if ( transB != MagmaNoTrans && transB != MagmaTrans && transB != MagmaConjTrans )
        info = -2;
    else if ( m < 0 )
        info = -3;
    else if ( n < 0 )
        info = -4;
    else if ( k < 0 )
        info = -5;
    else if ( transA == MagmaNoTrans ? ldda < m : ldda < k )
        info = -8;
    else if ( transB == MagmaNoTrans ? lddb < k : lddb < n )
        info = -10;
    else if ( lddc < m )
        info = -13;
    
    if( !(m == n  && n == k) ){
        printf("Only square sizes are supported\n");
        info = -1;
    }

    if( m > 32){
        printf("Only square sizes of up to 32 are supported\n");
        info = -1;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
    }

    if ( m <= 0 || n <= 0 || k <= 0 ) return;
    
    magma_int_t ntcol  = magma_get_dgemm_batched_ntcol( m );
    magma_int_t shmem  = ( SLDA(m)*m + SLDA(n)*n ) * sizeof(double);
                shmem *= ntcol;

    const int nblocks = magma_ceildiv(batchCount, ntcol);
    dim3 grid(nblocks, 1, 1);
    dim3 threads(m, m, ntcol);

    switch(m){
        case  1: dgemm_batched_smallsq_kernel< 1><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  2: dgemm_batched_smallsq_kernel< 2><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  3: dgemm_batched_smallsq_kernel< 3><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  4: dgemm_batched_smallsq_kernel< 4><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  5: dgemm_batched_smallsq_kernel< 5><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  6: dgemm_batched_smallsq_kernel< 6><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  7: dgemm_batched_smallsq_kernel< 7><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  8: dgemm_batched_smallsq_kernel< 8><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case  9: dgemm_batched_smallsq_kernel< 9><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 10: dgemm_batched_smallsq_kernel<10><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 11: dgemm_batched_smallsq_kernel<11><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 12: dgemm_batched_smallsq_kernel<12><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 13: dgemm_batched_smallsq_kernel<13><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 14: dgemm_batched_smallsq_kernel<14><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 15: dgemm_batched_smallsq_kernel<15><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 16: dgemm_batched_smallsq_kernel<16><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 17: dgemm_batched_smallsq_kernel<17><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 18: dgemm_batched_smallsq_kernel<18><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 19: dgemm_batched_smallsq_kernel<19><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 20: dgemm_batched_smallsq_kernel<20><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 21: dgemm_batched_smallsq_kernel<21><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 22: dgemm_batched_smallsq_kernel<22><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 23: dgemm_batched_smallsq_kernel<23><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 24: dgemm_batched_smallsq_kernel<24><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 25: dgemm_batched_smallsq_kernel<25><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 26: dgemm_batched_smallsq_kernel<26><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 27: dgemm_batched_smallsq_kernel<27><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 28: dgemm_batched_smallsq_kernel<28><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 29: dgemm_batched_smallsq_kernel<29><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 30: dgemm_batched_smallsq_kernel<30><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 31: dgemm_batched_smallsq_kernel<31><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        case 32: dgemm_batched_smallsq_kernel<32><<<grid, threads, shmem, queue->hip_stream()>>>(transA, transB, alpha, dA_array, ai, aj, ldda, dB_array, bi, bj, lddb, beta,  dC_array, ci, cj, lddc, batchCount); break;
        default:;
    }
}
