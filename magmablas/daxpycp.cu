#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @generated from magmablas/zaxpycp.cu, normal z -> d, Thu Oct  8 23:05:31 2020

*/
#include "magma_internal.h"

#define NB 64

/******************************************************************************/
// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
daxpycp_kernel(
    int m,
    double *r,
    double *x,
    const double *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_D_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


/***************************************************************************//**
    adds   x += r  --and--
    copies r = b
*******************************************************************************/
extern "C" void
magmablas_daxpycp(
    magma_int_t m,
    magmaDouble_ptr r,
    magmaDouble_ptr x,
    magmaDouble_const_ptr b,
    magma_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    daxpycp_kernel <<< grid, threads, 0, queue->hip_stream() >>> ( m, r, x, b );
}
