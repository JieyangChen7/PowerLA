#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Azzam Haidar
       @author Ahmad Abdelfattah

       @generated from magmablas/zgetf2_native_kernel.cu, normal z -> c, Thu Oct  8 23:05:37 2020
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "shuffle.cuh"
#include "sync.cuh"
#include "atomics.cuh"
#include "batched_kernel_param.h"

#define PRECISION_c

/**
    Purpose
    -------
    LU factorization of m-by-n matrix ( m >= n ).
    Each thread block caches an entire column in register.
    Thread blocks communicate and synchronize through global memory.
    Assumptions:
        1. dA is of size MxN such that N <= M.
        2. Thread block must be 1D, with TX multiple of 32 (warp size)
        3. TX must be >= n
        4. n must be less than the number of SMs on the GPU
**/

// =============================================================================
// init kernel
__global__ void
cgetf2_native_init_kernel( int n, int npages, magma_int_t *ipiv, int* update_flags)
{
    const int tx = threadIdx.x;
    if( tx < n){
        ipiv[ tx ] = 0;
    }
    if( tx < max(n,npages) ){
        update_flags[ tx ] = 0;
    }
}

// =============================================================================
// the main kernel
template<int TX, int NPAGES>
__global__ void
cgetf2_native_kernel( int m, int n,
                      magmaFloatComplex_ptr dA, int ldda,
                      volatile magma_int_t *ipiv, int gbstep,
                      volatile int* update_flag,
                      volatile magma_int_t *info)
{
#ifdef HAVE_CUBLAS
    const int tx  = threadIdx.x;
    const int bx = blockIdx.x;
    magmaFloatComplex rA[NPAGES] = {MAGMA_C_ZERO};
    magmaFloatComplex rx, rx_max;
    magmaFloatComplex_ptr da = dA;
    int rx_id, max_id, flag = 0;
    float  rx_abs = 0.0, rx_abs_max = 0.0;
    const int m_ = m-(NPAGES-1)*TX;
    if( bx >= n ) return;

    __shared__ magmaFloatComplex sx[ TX ];
    __shared__ float sabs[ TX ];
    __shared__ int smax_id[ TX ];
    __shared__ magmaFloatComplex sreg;

    // read
    dA += bx * ldda + tx;
    #pragma unroll
    for(int i = 0; i < NPAGES-1; i++){
        rA[i] = dA[ i * TX ];
    }
    if( tx <  m_){
        rA[NPAGES-1] = dA[ (NPAGES-1) * TX ];
    }

    // main loop
    #pragma unroll
    for(int i = 0; i < n; i++){
        // icamax and write pivot for the ith thread block
        if(bx == i){
            rx_max     = rx     = (tx < i) ? MAGMA_C_ZERO : rA[0];
            rx_abs_max = rx_abs = fabs(MAGMA_C_REAL(rx)) + fabs(MAGMA_C_IMAG(rx));
            max_id = rx_id = tx;
            #pragma unroll
            for(int j = 1; j < NPAGES; j++){
                rx = rA[j];
                rx_abs = fabs(MAGMA_C_REAL(rx)) + fabs(MAGMA_C_IMAG(rx));
                if ( rx_abs  > rx_abs_max ){
                    rx_max = rx;
                    rx_abs_max = rx_abs;
                    max_id = j * TX + tx;
                }
            }
            sx[ tx ] = rx_max;
            sabs[ tx ] = rx_abs_max;
            smax_id[ tx ] = max_id;
            __syncthreads();

            // let the first warp do the final reduction step
            if(tx < 32){
                #pragma unroll
                for(int j = 0; j < TX; j+= 32){
                    rx     = sx[ j + tx ];
                    rx_abs = sabs[ j + tx ];
                    rx_id  = smax_id[ j + tx ];
                    if ( rx_abs  > rx_abs_max ){
                        rx_max = rx;
                        rx_abs_max = rx_abs;
                        max_id = rx_id;
                    }
                }
                magmablas_syncwarp();
                sx[ tx ] = rx_max;
                sabs[ tx ] = rx_abs_max;
                smax_id[ tx ] = max_id;
                magmablas_syncwarp();
                #pragma unroll
                for(int j = 0; j < 32; j++){
                    rx     = sx[j];
                    rx_abs = sabs[j];
                    rx_id  = smax_id[j];
                    if ( rx_abs  > rx_abs_max ){
                        rx_abs_max = rx_abs;
                        rx_max = rx;
                        max_id = rx_id;
                    }
                }
            }

            if(tx == 0){
                sx[ 0 ] = rx_max;
                sabs[ 0 ] = rx_abs_max;
                smax_id[ 0 ] = max_id;
            }
            __syncthreads();
            rx_max = sx[ 0 ];
            rx_abs_max = sabs[ 0 ];
            max_id = smax_id[ 0 ];
            __syncthreads();

            // now every thread in the i^th block has the maximum
            if( tx == 0){
                if( rx_abs_max == MAGMA_D_ZERO){
                    magmablas_iatomic_exchange( (magma_int_t*)info, (magma_int_t)(max_id + gbstep + 1) );
                }
                magmablas_iatomic_exchange((magma_int_t*)&ipiv[i], (magma_int_t)(max_id+1) ); // fortran indexing
            }
            __syncthreads();
            if( rx_abs_max == MAGMA_D_ZERO )return;
        }
        else{ // other thread blocks are waiting
            if(tx == 0){
                max_id = 0;
                while( max_id == 0 ){
                    max_id = ipiv[i];
                };
                smax_id[ 0 ] = max_id;
            }
            __syncthreads();
            max_id = smax_id[ 0 ];
            max_id -= 1; // revert fortran indexing
            __syncthreads();
            if( (*info) != 0 ) return;
        }

        // swap
        // swap always happens between page 0 and page x
        // to avoid spilling rA to local memory, we use shared memory
        if( max_id != i){
            // all blocks swap in registers
            // for bx < i, the column is already written in memory,
            // but we have a copy in reg., so continue to swap in reg.,
            // and do one final write to memory
            #pragma unroll
            for(int j = 0; j < NPAGES; j++){
                if( j == (max_id/TX) ){
                    sx[ tx ] = rA[j];
                    __syncthreads();
                    if( tx == i ){
                        magmaFloatComplex tmp    = sx[ max_id%TX ];
                        sx[ max_id%TX ] = rA[0];
                        rA[0] = tmp;
                    }
                    __syncthreads();
                    if( tx == max_id%TX ){
                        rA[j] = sx[ tx ];
                    }
                    __syncthreads();
                }
            }
            //__syncthreads();
        }

        // the ith block does scal
        if(bx == i){
            magmaFloatComplex reg = MAGMA_C_DIV(MAGMA_C_ONE, rx_max );
            // scal
            if( tx > i ){
                rA[0] *= reg;
            }
            #pragma unroll
            for(int j = 1; j < NPAGES; j++){
                rA[j] *= reg;
            }
            // write column i to global memory
            #pragma unroll
            for(int j = 0; j < NPAGES-1; j++){
                dA[ j * TX ] = rA[j];
            }
            if( tx <  m_){
                dA[ (NPAGES-1) * TX ] = rA[NPAGES-1];
            }
            __threadfence(); __syncthreads(); // after cuda 9.0, both are needed, not sure why
            if(tx == 0) magmablas_iatomic_exchange( (int *)&update_flag[ i ], 1);
        }

        // thread blocks with ID larger than i perform ger
        if(bx > i){
            if( tx == i ){
                sreg = rA[0];
            }
            // wait for scal
            if( tx == 0){
                flag = 0;
                while( flag == 0 ){
                    flag = update_flag[ i ];
                };
            }
            __syncthreads();

            magmaFloatComplex reg = sreg;
            if( NPAGES == 1){
                if(tx > i && tx < m_){
                    rA[0] -= da[ i * ldda + tx ] * reg;
                }
            }else{
                if(tx > i){
                    rA[0] -= da[ i * ldda + tx ] * reg;
                }
            }
            #pragma unroll
            for(int j = 1; j < NPAGES-1; j++){
                rA[j] -= da[ i * ldda + j * TX + tx ] * reg;
            }
            if( NPAGES > 1){
                if( tx < m_ ){
                    rA[ NPAGES-1 ] -= da[ i * ldda + (NPAGES-1)*TX + tx ] * reg;
                }
            }
        }
    }

    // all blocks write their columns again except the last one
    if( bx < n-1 ){
        #pragma unroll
        for(int i = 0; i < NPAGES-1; i++){
            dA[ i * TX ] = rA[i];
        }
        if( tx <  m_){
            dA[ (NPAGES-1) * TX ] = rA[NPAGES-1];
        }
    }

#endif    // HAVE_CUBLAS
}

// =============================================================================
extern "C" magma_int_t
magma_cgetf2_native_fused(
    magma_int_t m, magma_int_t n,
    magmaFloatComplex_ptr dA, magma_int_t ldda,
    magma_int_t *ipiv, magma_int_t gbstep,
    magma_int_t *flags,
    magma_int_t *info, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    const magma_int_t ntx   = CGETF2_FUSED_NTH;

    if( m < n || m > CGETF2_FUSED_MAX_M ){
        arginfo = -1;
    }
    else if( n > magma_getdevice_multiprocessor_count() ){
        arginfo = -2;
    }
    else if( ldda < max(1, m) ){
        arginfo = -4;
    }

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }

    magma_int_t arch = magma_getdevice_arch();

    dim3 grid(n, 1, 1);
    dim3 threads(ntx, 1, 1);
    const magma_int_t npages = magma_ceildiv(m, ntx);
    // the kernel uses communication among thread blocks
    // as a safeguard, force one thread block per multiprocessor
    // by allocating more than half the shared memory
    magma_int_t shmem = magma_getdevice_shmem_block();
    shmem = (shmem / 2);
    int *update_flag = (int*) flags;    // update_flag is an int, not magma_int_t
    cgetf2_native_init_kernel<<< 1, max(n,npages), 0, queue->hip_stream() >>>( n, npages, ipiv, update_flag);
    // The case statement should cover up to ( xGETF2_CHAIN_MAX_M / ntx )
    switch(npages){
        case  1: cgetf2_native_kernel< ntx,  1><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  2: cgetf2_native_kernel< ntx,  2><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  3: cgetf2_native_kernel< ntx,  3><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  4: cgetf2_native_kernel< ntx,  4><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  5: cgetf2_native_kernel< ntx,  5><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  6: cgetf2_native_kernel< ntx,  6><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  7: cgetf2_native_kernel< ntx,  7><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  8: cgetf2_native_kernel< ntx,  8><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case  9: cgetf2_native_kernel< ntx,  9><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 10: cgetf2_native_kernel< ntx, 10><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 11: cgetf2_native_kernel< ntx, 11><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 12: cgetf2_native_kernel< ntx, 12><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 13: cgetf2_native_kernel< ntx, 13><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 14: cgetf2_native_kernel< ntx, 14><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 15: cgetf2_native_kernel< ntx, 15><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 16: cgetf2_native_kernel< ntx, 16><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 17: cgetf2_native_kernel< ntx, 17><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 18: cgetf2_native_kernel< ntx, 18><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 19: cgetf2_native_kernel< ntx, 19><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 20: cgetf2_native_kernel< ntx, 20><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        #if defined(PRECISION_s) || defined(PRECISION_d)
        case 21: cgetf2_native_kernel< ntx, 21><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 22: cgetf2_native_kernel< ntx, 22><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 23: cgetf2_native_kernel< ntx, 23><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 24: cgetf2_native_kernel< ntx, 24><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 25: cgetf2_native_kernel< ntx, 25><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 26: cgetf2_native_kernel< ntx, 26><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 27: cgetf2_native_kernel< ntx, 27><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 28: cgetf2_native_kernel< ntx, 28><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 29: cgetf2_native_kernel< ntx, 29><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 30: cgetf2_native_kernel< ntx, 30><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 31: cgetf2_native_kernel< ntx, 31><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 32: cgetf2_native_kernel< ntx, 32><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 33: cgetf2_native_kernel< ntx, 33><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 34: cgetf2_native_kernel< ntx, 34><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 35: cgetf2_native_kernel< ntx, 35><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 36: cgetf2_native_kernel< ntx, 36><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 37: cgetf2_native_kernel< ntx, 37><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 38: cgetf2_native_kernel< ntx, 38><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 39: cgetf2_native_kernel< ntx, 39><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 40: cgetf2_native_kernel< ntx, 40><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 41: cgetf2_native_kernel< ntx, 41><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 42: cgetf2_native_kernel< ntx, 42><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 43: cgetf2_native_kernel< ntx, 43><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 44: cgetf2_native_kernel< ntx, 44><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 45: cgetf2_native_kernel< ntx, 45><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 46: cgetf2_native_kernel< ntx, 46><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        #endif // defined(PRECISION_s) || defined(PRECISION_d)
        #if defined(PRECISION_s)
        case 47: cgetf2_native_kernel< ntx, 47><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 48: cgetf2_native_kernel< ntx, 48><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 49: cgetf2_native_kernel< ntx, 49><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 50: cgetf2_native_kernel< ntx, 50><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 51: cgetf2_native_kernel< ntx, 51><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 52: cgetf2_native_kernel< ntx, 52><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 53: cgetf2_native_kernel< ntx, 53><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 54: cgetf2_native_kernel< ntx, 54><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 55: cgetf2_native_kernel< ntx, 55><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 56: cgetf2_native_kernel< ntx, 56><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 57: cgetf2_native_kernel< ntx, 57><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 58: cgetf2_native_kernel< ntx, 58><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 59: cgetf2_native_kernel< ntx, 59><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 60: cgetf2_native_kernel< ntx, 60><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 61: cgetf2_native_kernel< ntx, 61><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 62: cgetf2_native_kernel< ntx, 62><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 63: cgetf2_native_kernel< ntx, 63><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 64: cgetf2_native_kernel< ntx, 64><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 65: cgetf2_native_kernel< ntx, 65><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 66: cgetf2_native_kernel< ntx, 66><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 67: cgetf2_native_kernel< ntx, 67><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 68: cgetf2_native_kernel< ntx, 68><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 69: cgetf2_native_kernel< ntx, 69><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 70: cgetf2_native_kernel< ntx, 70><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 71: cgetf2_native_kernel< ntx, 71><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 72: cgetf2_native_kernel< ntx, 72><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 73: cgetf2_native_kernel< ntx, 73><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 74: cgetf2_native_kernel< ntx, 74><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 75: cgetf2_native_kernel< ntx, 75><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 76: cgetf2_native_kernel< ntx, 76><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 77: cgetf2_native_kernel< ntx, 77><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 78: cgetf2_native_kernel< ntx, 78><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 79: cgetf2_native_kernel< ntx, 79><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        case 80: cgetf2_native_kernel< ntx, 80><<<grid, threads, shmem, queue->hip_stream() >>>( m, n, dA, ldda, ipiv, gbstep, update_flag, info); break;
        #endif // defined(PRECISION_s)
        default: printf("size not supported \n");
    }
    return 0;
}
