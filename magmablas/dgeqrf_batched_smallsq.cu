#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Ahmad Abdelfattah
       @author Azzam Haidar

       @generated from magmablas/zgeqrf_batched_smallsq.cu, normal z -> d, Thu Oct  8 23:05:37 2020
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "sync.cuh"
#include "batched_kernel_param.h"

#define SLDA(N)    ( (N==15||N==23||N==31)? (N+2) : (N+1) )
extern __shared__ double zdata[];
template<int N>
__global__ void
dgeqrf_batched_sq1d_reg_kernel( 
    double **dA_array, magma_int_t ldda,
    double **dtau_array, magma_int_t *info_array, 
    magma_int_t batchCount)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y; 
    const int batchid = blockIdx.x * blockDim.y + ty;
    if(batchid >= batchCount) return;
    if(tx >= N) return;
    
    const int slda  = SLDA(N);
    double* dA   = dA_array[batchid];
    double* dtau = dtau_array[batchid];
    magma_int_t* info = &info_array[batchid];
    // shared memory pointers
    double* sA = (double*)(zdata + ty * slda * N);
    double* sdw = (double*)(zdata + blockDim.y * slda * N);
    sdw += ty * N;

    double rA[N] = {MAGMA_D_ZERO};
    double alpha, tau, tmp, zsum, scale = MAGMA_D_ZERO;
    double sum = MAGMA_D_ZERO, norm = MAGMA_D_ZERO, beta; 

    if( tx == 0 ){
        (*info) = 0;
    }

    // init tau
    dtau[tx] = MAGMA_D_ZERO;
    // read 
    #pragma unroll
    for(int i = 0; i < N; i++){
        rA[i] = dA[ i * ldda + tx ];
    }
    
    #pragma unroll
    for(int i = 0; i < N-1; i++){
        sA[ i * slda + tx] = rA[i];
        sdw[tx] = ( MAGMA_D_REAL(rA[i]) * MAGMA_D_REAL(rA[i]) + MAGMA_D_IMAG(rA[i]) * MAGMA_D_IMAG(rA[i]) );
        magmablas_syncwarp();
        alpha = sA[i * slda + i];
        sum = MAGMA_D_ZERO; 
        #pragma unroll
        for(int j = i; j < N; j++){
            sum += sdw[j];
        }
        norm = sqrt(sum);
        beta = -copysign(norm, real(alpha));
        scale = MAGMA_D_DIV( MAGMA_D_ONE,  alpha - MAGMA_D_MAKE(beta, 0));
        tau = MAGMA_D_MAKE( (beta - real(alpha)) / beta, -imag(alpha) / beta );

        if(tx == i){
            dtau[i] = tau;
        }
        
        tmp = (tx == i)? MAGMA_D_MAKE(beta, MAGMA_D_ZERO) : rA[i] * scale;
        
        if(tx >= i){
            rA[i] = tmp;
        }
        
        dA[ i * ldda + tx ] = rA[i];
        rA[i] = (tx == i) ? MAGMA_D_ONE  : rA[i]; 
        rA[i] = (tx < i ) ? MAGMA_D_ZERO : rA[i];
        tmp = MAGMA_D_CONJ( rA[i] ) * MAGMA_D_CONJ( tau );
        
        magmablas_syncwarp();
        #pragma unroll
        for(int j = i+1; j < N; j++){
            sA[j * slda + tx] = rA[j] * tmp;
        }
        magmablas_syncwarp();

        zsum = MAGMA_D_ZERO;
        #pragma unroll
        for(int j = i; j < N; j++){
            zsum += sA[tx * slda + j];
        }
        sA[tx * slda + N] = zsum;
        magmablas_syncwarp();
        
        #pragma unroll
        for(int j = i+1; j < N; j++){
            rA[j] -= rA[i] * sA[j * slda + N];
        }
        magmablas_syncwarp();
    }
    // write the last column
    dA[ (N-1) * ldda + tx ] = rA[N-1];
}

/***************************************************************************//**
    Purpose
    -------
    DGEQRF computes a QR factorization of a real M-by-N matrix A:
    A = Q * R.
    
    This is a batched version of the routine, and works only for small 
    square matrices of size up to 32.
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The size of the matrix A.  N >= 0.

    @param[in,out]
    dA_array Array of pointers, dimension (batchCount).
             Each is a DOUBLE PRECISION array on the GPU, dimension (LDDA,N)
             On entry, the M-by-N matrix A.
             On exit, the elements on and above the diagonal of the array
             contain the min(M,N)-by-N upper trapezoidal matrix R (R is
             upper triangular if m >= n); the elements below the diagonal,
             with the array TAU, represent the orthogonal matrix Q as a
             product of min(m,n) elementary reflectors (see Further
             Details).

    @param[in]
    ldda     INTEGER
             The leading dimension of the array dA.  LDDA >= max(1,M).
             To benefit from coalescent memory accesses LDDA must be
             divisible by 16.

    @param[out]
    dtau_array Array of pointers, dimension (batchCount).
             Each is a DOUBLE PRECISION array, dimension (min(M,N))
             The scalar factors of the elementary reflectors (see Further
             Details).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    Further Details
    ---------------
    The matrix Q is represented as a product of elementary reflectors

       Q = H(1) H(2) . . . H(k), where k = min(m,n).

    Each H(i) has the form

       H(i) = I - tau * v * v'

    where tau is a real scalar, and v is a real vector with
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),
    and tau in TAU(i).

    @ingroup magma_geqrf_batched
*******************************************************************************/
extern "C" magma_int_t 
magma_dgeqrf_batched_smallsq( 
    magma_int_t n,  
    double** dA_array, magma_int_t ldda, 
    double **dtau_array, magma_int_t* info_array, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    magma_int_t m = n;
    if( (m < 0) || ( m > 32 ) ){
        arginfo = -1;
    }

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }
    
    if( m == 0 || n == 0) return 0;
    
    const magma_int_t ntcol = magma_get_dgeqrf_batched_ntcol(m, n);
    
    magma_int_t shmem = ( SLDA(m) * m * sizeof(double) );
    shmem            += ( m * sizeof(double) );
    shmem            *= ntcol;
    magma_int_t nth = magma_ceilpow2(m);
    magma_int_t gridx = magma_ceildiv(batchCount, ntcol);
    dim3 grid(gridx, 1, 1);
    dim3 threads(nth, ntcol, 1);
    
    switch(m){
        
        case  1: dgeqrf_batched_sq1d_reg_kernel< 1><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  2: dgeqrf_batched_sq1d_reg_kernel< 2><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  3: dgeqrf_batched_sq1d_reg_kernel< 3><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  4: dgeqrf_batched_sq1d_reg_kernel< 4><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  5: dgeqrf_batched_sq1d_reg_kernel< 5><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  6: dgeqrf_batched_sq1d_reg_kernel< 6><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  7: dgeqrf_batched_sq1d_reg_kernel< 7><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  8: dgeqrf_batched_sq1d_reg_kernel< 8><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  9: dgeqrf_batched_sq1d_reg_kernel< 9><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 10: dgeqrf_batched_sq1d_reg_kernel<10><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 11: dgeqrf_batched_sq1d_reg_kernel<11><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 12: dgeqrf_batched_sq1d_reg_kernel<12><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 13: dgeqrf_batched_sq1d_reg_kernel<13><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 14: dgeqrf_batched_sq1d_reg_kernel<14><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 15: dgeqrf_batched_sq1d_reg_kernel<15><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 16: dgeqrf_batched_sq1d_reg_kernel<16><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 17: dgeqrf_batched_sq1d_reg_kernel<17><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 18: dgeqrf_batched_sq1d_reg_kernel<18><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 19: dgeqrf_batched_sq1d_reg_kernel<19><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 20: dgeqrf_batched_sq1d_reg_kernel<20><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 21: dgeqrf_batched_sq1d_reg_kernel<21><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 22: dgeqrf_batched_sq1d_reg_kernel<22><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 23: dgeqrf_batched_sq1d_reg_kernel<23><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 24: dgeqrf_batched_sq1d_reg_kernel<24><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 25: dgeqrf_batched_sq1d_reg_kernel<25><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 26: dgeqrf_batched_sq1d_reg_kernel<26><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 27: dgeqrf_batched_sq1d_reg_kernel<27><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 28: dgeqrf_batched_sq1d_reg_kernel<28><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 29: dgeqrf_batched_sq1d_reg_kernel<29><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 30: dgeqrf_batched_sq1d_reg_kernel<30><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 31: dgeqrf_batched_sq1d_reg_kernel<31><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 32: dgeqrf_batched_sq1d_reg_kernel<32><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        default: printf("error: size %lld is not supported\n", (long long) m);
    }
    return arginfo;
}
