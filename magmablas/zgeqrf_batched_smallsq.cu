#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.5.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date October 2020

       @author Ahmad Abdelfattah
       @author Azzam Haidar

       @precisions normal z -> s d c
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "sync.cuh"
#include "batched_kernel_param.h"

#define SLDA(N)    ( (N==15||N==23||N==31)? (N+2) : (N+1) )
extern __shared__ magmaDoubleComplex zdata[];
template<int N>
__global__ void
zgeqrf_batched_sq1d_reg_kernel( 
    magmaDoubleComplex **dA_array, magma_int_t ldda,
    magmaDoubleComplex **dtau_array, magma_int_t *info_array, 
    magma_int_t batchCount)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y; 
    const int batchid = blockIdx.x * blockDim.y + ty;
    if(batchid >= batchCount) return;
    if(tx >= N) return;
    
    const int slda  = SLDA(N);
    magmaDoubleComplex* dA   = dA_array[batchid];
    magmaDoubleComplex* dtau = dtau_array[batchid];
    magma_int_t* info = &info_array[batchid];
    // shared memory pointers
    magmaDoubleComplex* sA = (magmaDoubleComplex*)(zdata + ty * slda * N);
    double* sdw = (double*)(zdata + blockDim.y * slda * N);
    sdw += ty * N;

    magmaDoubleComplex rA[N] = {MAGMA_Z_ZERO};
    magmaDoubleComplex alpha, tau, tmp, zsum, scale = MAGMA_Z_ZERO;
    double sum = MAGMA_D_ZERO, norm = MAGMA_D_ZERO, beta; 

    if( tx == 0 ){
        (*info) = 0;
    }

    // init tau
    dtau[tx] = MAGMA_Z_ZERO;
    // read 
    #pragma unroll
    for(int i = 0; i < N; i++){
        rA[i] = dA[ i * ldda + tx ];
    }
    
    #pragma unroll
    for(int i = 0; i < N-1; i++){
        sA[ i * slda + tx] = rA[i];
        sdw[tx] = ( MAGMA_Z_REAL(rA[i]) * MAGMA_Z_REAL(rA[i]) + MAGMA_Z_IMAG(rA[i]) * MAGMA_Z_IMAG(rA[i]) );
        magmablas_syncwarp();
        alpha = sA[i * slda + i];
        sum = MAGMA_D_ZERO; 
        #pragma unroll
        for(int j = i; j < N; j++){
            sum += sdw[j];
        }
        norm = sqrt(sum);
        beta = -copysign(norm, real(alpha));
        scale = MAGMA_Z_DIV( MAGMA_Z_ONE,  alpha - MAGMA_Z_MAKE(beta, 0));
        tau = MAGMA_Z_MAKE( (beta - real(alpha)) / beta, -imag(alpha) / beta );

        if(tx == i){
            dtau[i] = tau;
        }
        
        tmp = (tx == i)? MAGMA_Z_MAKE(beta, MAGMA_D_ZERO) : rA[i] * scale;
        
        if(tx >= i){
            rA[i] = tmp;
        }
        
        dA[ i * ldda + tx ] = rA[i];
        rA[i] = (tx == i) ? MAGMA_Z_ONE  : rA[i]; 
        rA[i] = (tx < i ) ? MAGMA_Z_ZERO : rA[i];
        tmp = MAGMA_Z_CONJ( rA[i] ) * MAGMA_Z_CONJ( tau );
        
        magmablas_syncwarp();
        #pragma unroll
        for(int j = i+1; j < N; j++){
            sA[j * slda + tx] = rA[j] * tmp;
        }
        magmablas_syncwarp();

        zsum = MAGMA_Z_ZERO;
        #pragma unroll
        for(int j = i; j < N; j++){
            zsum += sA[tx * slda + j];
        }
        sA[tx * slda + N] = zsum;
        magmablas_syncwarp();
        
        #pragma unroll
        for(int j = i+1; j < N; j++){
            rA[j] -= rA[i] * sA[j * slda + N];
        }
        magmablas_syncwarp();
    }
    // write the last column
    dA[ (N-1) * ldda + tx ] = rA[N-1];
}

/***************************************************************************//**
    Purpose
    -------
    ZGEQRF computes a QR factorization of a complex M-by-N matrix A:
    A = Q * R.
    
    This is a batched version of the routine, and works only for small 
    square matrices of size up to 32.
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The size of the matrix A.  N >= 0.

    @param[in,out]
    dA_array Array of pointers, dimension (batchCount).
             Each is a COMPLEX_16 array on the GPU, dimension (LDDA,N)
             On entry, the M-by-N matrix A.
             On exit, the elements on and above the diagonal of the array
             contain the min(M,N)-by-N upper trapezoidal matrix R (R is
             upper triangular if m >= n); the elements below the diagonal,
             with the array TAU, represent the orthogonal matrix Q as a
             product of min(m,n) elementary reflectors (see Further
             Details).

    @param[in]
    ldda     INTEGER
             The leading dimension of the array dA.  LDDA >= max(1,M).
             To benefit from coalescent memory accesses LDDA must be
             divisible by 16.

    @param[out]
    dtau_array Array of pointers, dimension (batchCount).
             Each is a COMPLEX_16 array, dimension (min(M,N))
             The scalar factors of the elementary reflectors (see Further
             Details).

    @param[out]
    info_array  Array of INTEGERs, dimension (batchCount), for corresponding matrices.
      -     = 0:  successful exit

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    Further Details
    ---------------
    The matrix Q is represented as a product of elementary reflectors

       Q = H(1) H(2) . . . H(k), where k = min(m,n).

    Each H(i) has the form

       H(i) = I - tau * v * v'

    where tau is a complex scalar, and v is a complex vector with
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),
    and tau in TAU(i).

    @ingroup magma_geqrf_batched
*******************************************************************************/
extern "C" magma_int_t 
magma_zgeqrf_batched_smallsq( 
    magma_int_t n,  
    magmaDoubleComplex** dA_array, magma_int_t ldda, 
    magmaDoubleComplex **dtau_array, magma_int_t* info_array, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    magma_int_t m = n;
    if( (m < 0) || ( m > 32 ) ){
        arginfo = -1;
    }

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }
    
    if( m == 0 || n == 0) return 0;
    
    const magma_int_t ntcol = magma_get_zgeqrf_batched_ntcol(m, n);
    
    magma_int_t shmem = ( SLDA(m) * m * sizeof(magmaDoubleComplex) );
    shmem            += ( m * sizeof(double) );
    shmem            *= ntcol;
    magma_int_t nth = magma_ceilpow2(m);
    magma_int_t gridx = magma_ceildiv(batchCount, ntcol);
    dim3 grid(gridx, 1, 1);
    dim3 threads(nth, ntcol, 1);
    
    switch(m){
        
        case  1: zgeqrf_batched_sq1d_reg_kernel< 1><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  2: zgeqrf_batched_sq1d_reg_kernel< 2><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  3: zgeqrf_batched_sq1d_reg_kernel< 3><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  4: zgeqrf_batched_sq1d_reg_kernel< 4><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  5: zgeqrf_batched_sq1d_reg_kernel< 5><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  6: zgeqrf_batched_sq1d_reg_kernel< 6><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  7: zgeqrf_batched_sq1d_reg_kernel< 7><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  8: zgeqrf_batched_sq1d_reg_kernel< 8><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case  9: zgeqrf_batched_sq1d_reg_kernel< 9><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 10: zgeqrf_batched_sq1d_reg_kernel<10><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 11: zgeqrf_batched_sq1d_reg_kernel<11><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 12: zgeqrf_batched_sq1d_reg_kernel<12><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 13: zgeqrf_batched_sq1d_reg_kernel<13><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 14: zgeqrf_batched_sq1d_reg_kernel<14><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 15: zgeqrf_batched_sq1d_reg_kernel<15><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 16: zgeqrf_batched_sq1d_reg_kernel<16><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 17: zgeqrf_batched_sq1d_reg_kernel<17><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 18: zgeqrf_batched_sq1d_reg_kernel<18><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 19: zgeqrf_batched_sq1d_reg_kernel<19><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 20: zgeqrf_batched_sq1d_reg_kernel<20><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 21: zgeqrf_batched_sq1d_reg_kernel<21><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 22: zgeqrf_batched_sq1d_reg_kernel<22><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 23: zgeqrf_batched_sq1d_reg_kernel<23><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 24: zgeqrf_batched_sq1d_reg_kernel<24><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 25: zgeqrf_batched_sq1d_reg_kernel<25><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 26: zgeqrf_batched_sq1d_reg_kernel<26><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 27: zgeqrf_batched_sq1d_reg_kernel<27><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 28: zgeqrf_batched_sq1d_reg_kernel<28><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 29: zgeqrf_batched_sq1d_reg_kernel<29><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 30: zgeqrf_batched_sq1d_reg_kernel<30><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 31: zgeqrf_batched_sq1d_reg_kernel<31><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        case 32: zgeqrf_batched_sq1d_reg_kernel<32><<<grid, threads, shmem, queue->hip_stream()>>>(dA_array, ldda, dtau_array, info_array, batchCount); break;
        default: printf("error: size %lld is not supported\n", (long long) m);
    }
    return arginfo;
}
